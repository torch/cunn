#include "hip/hip_runtime.h"
#include "utils.h"

struct expupdateOutput_functor
{
  __host__ __device__ float operator()(const float& input) const
  {
    return exp(input);
  }
};

static int cunn_Exp_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  long size = THCudaTensor_nElement(state, input);

  input = THCudaTensor_newContiguous(state, input);

  THCudaTensor_resizeAs(state, output, input);

  thrust::device_ptr<float> output_data(THCudaTensor_data(state, output));
  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::transform(input_data, input_data+size, output_data, expupdateOutput_functor());

  THCudaTensor_free(state, input);
  return 1;
}

struct expupdateGradInput_functor
{
  __host__ __device__ float operator()(const float& output, const float& gradOutput) const
  {
    return gradOutput * output;
  }
};

static int cunn_Exp_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  long size = THCudaTensor_nElement(state, output);

  gradOutput = THCudaTensor_newContiguous(state, gradOutput);

  THCudaTensor_resizeAs(state, gradInput, output);

  thrust::device_ptr<float> output_data(THCudaTensor_data(state, output));
  thrust::device_ptr<float> gradOutput_data(THCudaTensor_data(state, gradOutput));
  thrust::device_ptr<float> gradInput_data(THCudaTensor_data(state, gradInput));
  thrust::transform(output_data, output_data+size, gradOutput_data, gradInput_data, expupdateGradInput_functor());

  THCudaTensor_free(state, gradOutput);
  return 1;
}

static const struct luaL_Reg cunn_Exp__ [] = {
  {"Exp_updateOutput", cunn_Exp_updateOutput},
  {"Exp_updateGradInput", cunn_Exp_updateGradInput},
  {NULL, NULL}
};

static void cunn_Exp_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_Exp__, "nn");
  lua_pop(L,1);
}
