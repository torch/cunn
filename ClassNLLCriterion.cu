#include "hip/hip_runtime.h"
/**
 * Copyright 2014 Facebook
 */

#include "utils.h"
#include<assert.h>

static const int NTHREADS = 32;

__global__ void cunn_ClassNLLCriterion_updateOutput_kernel1(float *output,
                                                            float *input,
                                                            float *target,
                                                            int ntarget) {
  assert(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0);

  // TODO: T4951791 Reuse code between updateOutput_kernel1 and
  // updateOutput_kernel.
  // Verify whether `register` does anything here.
  register int i, t;
  for (i = 0; i < ntarget; i++) {
    t = target[i] - 1;
    if (t >= 0)
      *output = -input[t];
  }
}

__global__ void cunn_ClassNLLCriterion_updateOutput_kernel(float *output,
                                                           float *input,
                                                           float *target,
                                                           int nframe,
                                                           int ndim,
                                                           int sizeAverage,
                                                           int ntarget) {
  __shared__ float shInputs[NTHREADS];
  // Verify whether `register` does anything here.
  register int i, j, t;

  shInputs[threadIdx.x] = .0;
  for (i = threadIdx.x; i < nframe; i += NTHREADS) {
    for (j = 0; j < ntarget; ++j) {
      t = (int)target[i * ntarget + j] - 1;
      if (t >= 0)
        shInputs[threadIdx.x] += input[i * ndim + t];
    }
  }
  __syncthreads();

  // TODO: T4951791 Reuse code between updateOutput_kernel1 and
  // updateOutput_kernel
  if (threadIdx.x == 0) {
    *output = .0;
    for (i = 0; i < NTHREADS; ++i)
      *output += shInputs[i];
    if (sizeAverage)
      *output /= nframe;
    *output = -(*output);
  }
}

__global__ void cunn_ClassNLLCriterion_updateGradInput_kernel(float *gradInput,
                                                              float *target,
                                                              int nframe,
                                                              int ndim,
                                                              float grad,
                                                              int ntarget) {
  register int i, j, t;
  for (i = threadIdx.x; i < nframe; i += NTHREADS) {
    for (j = 0; j < ntarget; ++j) {
      t = (int)target[i * ntarget + j] - 1;
      if (t >= 0)
        gradInput[i * ndim + t] = grad;
    }
  }
}

static int cunn_ClassNLLCriterion_updateOutput(lua_State *L) {
  THCState *state = getCutorchState(L);
  THCudaTensor *input =
      (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  input = THCudaTensor_newContiguous(state, input);
  float *input_data = THCudaTensor_data(state, input);

  THCudaTensor *target =
      (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  target = THCudaTensor_newContiguous(state, target);
  float *target_data = THCudaTensor_data(state, target);
  int ntarget = 1;
  if (target->nDimension > 1)
    ntarget = target->size[1];

  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(
      L, 1, "outputTensor", "torch.CudaTensor");
  output = THCudaTensor_newContiguous(state, output);
  float *output_data = THCudaTensor_data(state, output);

  if (input->nDimension == 1) {
    cunn_ClassNLLCriterion_updateOutput_kernel1 << <1, 1>>>
        (output_data, input_data, target_data, ntarget);
  } else if (input->nDimension == 2) {
    dim3 blocks(1);
    dim3 threads(NTHREADS);
    int sizeAverage = luaT_getfieldcheckboolean(L, 1, "sizeAverage");
    cunn_ClassNLLCriterion_updateOutput_kernel <<<blocks, threads>>>
        (output_data,
         input_data,
         target_data,
         input->size[0],
         input->size[1],
         sizeAverage,
         ntarget);
  } else
    THArgCheck(0, 2, "vector or matrix expected");

  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  THCudaTensor_free(state, output);
  THCudaTensor_free(state, target);
  THCudaTensor_free(state, input);

  return 1;
}

static int cunn_ClassNLLCriterion_updateGradInput(lua_State *L) {
  THCState *state = getCutorchState(L);

  THCudaTensor *input =
      (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  input = THCudaTensor_newContiguous(state, input);

  THCudaTensor *target =
      (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  target = THCudaTensor_newContiguous(state, target);
  float *target_data = THCudaTensor_data(state, target);
  int ntarget = 1;
  if (target->nDimension > 1)
    ntarget = target->size[1];

  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(
      L, 1, "gradInput", "torch.CudaTensor");
  gradInput = THCudaTensor_newContiguous(state, gradInput);
  float *gradInput_data = THCudaTensor_data(state, gradInput);

  float grad = -1.0;
  if (input->nDimension == 1) {
    if (ntarget > 1)
      THArgCheck(0, 2, "multi-target not implemented");
    float tid;
    hipMemcpy(&tid, target_data, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(gradInput_data + (int)tid - 1,
               &grad,
               sizeof(float),
               hipMemcpyHostToDevice);
  } else if (input->nDimension == 2) {
    int nframe = input->size[0];
    int ndim = input->size[1];
    int sizeAverage = luaT_getfieldcheckboolean(L, 1, "sizeAverage");
    if (sizeAverage)
      grad /= nframe;
    dim3 blocks(1);
    dim3 threads(NTHREADS);
    cunn_ClassNLLCriterion_updateGradInput_kernel <<<blocks, threads>>>
        (gradInput_data, target_data, nframe, ndim, grad, ntarget);
  } else
    THArgCheck(0, 2, "vector or matrix expected");

  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess)
    THError(hipGetErrorString(errcode));

  THCudaTensor_free(state, gradInput);
  THCudaTensor_free(state, target);
  THCudaTensor_free(state, input);

  return 1;
}

static const struct luaL_Reg cunn_ClassNLLCriterion__[] = {
    {"ClassNLLCriterion_updateOutput", cunn_ClassNLLCriterion_updateOutput},
    {"ClassNLLCriterion_updateGradInput",
     cunn_ClassNLLCriterion_updateGradInput},
    {NULL, NULL}};

void cunn_ClassNLLCriterion_init(lua_State *L) {
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_ClassNLLCriterion__, "nn");
  lua_pop(L, 1);
}
