#include "hip/hip_runtime.h"
#include "utils.h"

struct tanhupdateOutput_functor
{
  __device__ void operator()(float* output, const float* input) const
  {
    *output = tanh(*input);
  }
};

static int cunn_Tanh_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

  THCudaTensor_resizeAs(state, output, input);
  THCudaTensor_pointwiseApply2(state, output, input, tanhupdateOutput_functor());
  return 1;
}

struct tanhupdateGradInput_functor
{
  __device__ void operator()(float* gradInput, const float* output, const float* gradOutput) const
  {
    *gradInput = *gradOutput * (1 - *output * *output);
  }
};

static int cunn_Tanh_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

  THCudaTensor_resizeAs(state, gradInput, output);
  THCudaTensor_pointwiseApply3(state, gradInput, output, gradOutput, tanhupdateGradInput_functor());
  return 1;
}

static const struct luaL_Reg cunn_Tanh__ [] = {
  {"Tanh_updateOutput", cunn_Tanh_updateOutput},
  {"Tanh_updateGradInput", cunn_Tanh_updateGradInput},
  {NULL, NULL}
};

static void cunn_Tanh_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_Tanh__, "nn");
  lua_pop(L,1);
}
