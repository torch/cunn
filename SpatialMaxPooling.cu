#include "hip/hip_runtime.h"
#include "utils.h"

#define CUDA_MAX_THREADS 1024   // this is safe, in reality 256 is our limit

/*
 * Description:
 *    this function maxpools an input 4D tensor along dimensions 2 and 3
 *    4D input, 4D output, 4D argmax x and y 
 */
__global__ void maxpool(float *input, float *output, float *indices_x, float *indices_y,
                        int input_n, int input_h, int input_w,
                        int kH, int kW, int dH, int dW)
{
  // iterators
  int xx, yy;

  // output size
  const int output_w = (input_w - kW) / dW + 1;
  const int output_h = (input_h - kH) / dH + 1;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;
  //int k = blockIdx.x % input_n;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  const int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  const int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  output = output + o*output_w*output_h;
  input = input + i*input_w*input_h;
  indices_x = indices_x + o*output_w*output_h;
  indices_y = indices_y + o*output_w*output_h;

  // For all output pixels...
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {
    for(xx = xx_start; xx < xx_end; xx+=xx_step) {
      // Compute the mean of the input image...
      float *ptr_input = input + yy*dH*input_w + xx*dW;
      float *ptr_output = output + yy*output_w + xx;
      float *ptr_ind_x = indices_x + yy*output_w + xx;
      float *ptr_ind_y = indices_y + yy*output_w + xx;
      int argmax_x = -1;
      int argmax_y = -1;
      float max = -FLT_MAX;
      int kx, ky;
      for(ky = 0; ky < kH; ky++) {
        for(kx = 0; kx < kW; kx++) {
          float val = ptr_input[kx];
          if (val > max) {
            max = val;
            argmax_x = kx;
            argmax_y = ky;
          } 
        }
        ptr_input += input_w; // next input line
      }
      // Update output and argmax
      *ptr_output = max;
      *ptr_ind_x = argmax_x + 1;
      *ptr_ind_y = argmax_y + 1;
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from weight and gradOutput
 */
__global__ void maxgradinput(float *gradInput, float *gradOutput, float *indices_x, float *indices_y,
                             int input_n, int input_h, int input_w,
                             int kH, int kW, int dH, int dW)
{
  // iterators
  int xx, yy;

  // output size
  int output_w = (input_w - kW) / dW + 1;
  int output_h = (input_h - kH) / dH + 1;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;
  //int k = blockIdx.x % input_n;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o*output_w*output_h;
  gradInput = gradInput + i*input_w*input_h;
  indices_x = indices_x + o*output_w*output_h;
  indices_y = indices_y + o*output_w*output_h;

  // compute gradInput
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {
    for(xx = xx_start; xx < xx_end; xx+=xx_step) {
      float *ptr_gradInput = gradInput + yy*dH*input_w + xx*dW;
      float *ptr_gradOutput = gradOutput + yy*output_w + xx;
      float *ptr_ind_x = indices_x + yy*output_w + xx;
      float *ptr_ind_y = indices_y + yy*output_w + xx;
      float z = *ptr_gradOutput;

      int argmax_x = (*ptr_ind_x)-1;
      int argmax_y = (*ptr_ind_y)-1;

      ptr_gradInput[argmax_x + argmax_y*input_w] += z;
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from weight and gradOutput
 *    when kH != dH or kW != dW (uses atomic add)
 */
__global__ void atomicmaxgradinput(
  float *gradInput, float *gradOutput, float *indices_x, float *indices_y,
  int input_n, int input_h, int input_w, int kH, int kW, int dH, int dW
)
{
  // iterators
  int xx, yy;

  // output size
  int output_w = (input_w - kW) / dW + 1;
  int output_h = (input_h - kH) / dH + 1;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;
  //int k = blockIdx.x % input_n;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o*output_w*output_h;
  gradInput = gradInput + i*input_w*input_h;
  indices_x = indices_x + o*output_w*output_h;
  indices_y = indices_y + o*output_w*output_h;

  // compute gradInput
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {
    for(xx = xx_start; xx < xx_end; xx+=xx_step) {
      float *ptr_gradInput = gradInput + yy*dH*input_w + xx*dW;
      float *ptr_gradOutput = gradOutput + yy*output_w + xx;
      float *ptr_ind_x = indices_x + yy*output_w + xx;
      float *ptr_ind_y = indices_y + yy*output_w + xx;
      float z = *ptr_gradOutput;

      int argmax_x = (*ptr_ind_x)-1;
      int argmax_y = (*ptr_ind_y)-1;

      // atomic add since different threads could update same variable
      atomicAdd(&(ptr_gradInput[argmax_x + argmax_y*input_w]), z);
    }
  }
}

static int cunn_SpatialMaxPooling_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");

  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *indices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "indices", "torch.CudaTensor");

  float *indices_data;
  float *output_data;
  float *input_data;

  luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  if (input->nDimension == 3) {
    long nInputCols = input->size[2];
    long nInputRows = input->size[1];
    long nInputPlane = input->size[0];
    long nOutputCols = (nInputCols - kW) / dW + 1;
    long nOutputRows = (nInputRows - kH) / dH + 1;

    luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

    input = THCudaTensor_newContiguous(state, input);
    input_data = THCudaTensor_data(state, input);

    THCudaTensor_resize3d(state, output, nInputPlane, nOutputRows, nOutputCols);
    THCudaTensor_resize4d(state, indices, 2, nInputPlane, nOutputRows, nOutputCols);

    indices_data = THCudaTensor_data(state, indices);
    output_data = THCudaTensor_data(state, output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    // run maxpool kernel
    maxpool <<<blocks, threads>>> (input_data, output_data,
                                   indices_data+nInputPlane*nOutputCols*nOutputRows, indices_data,
                                   nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
  } else {
    long nInputCols = input->size[3];
    long nInputRows = input->size[2];
    long nInputPlane = input->size[1];
    long nbatch = input->size[0];
    long nOutputCols = (nInputCols - kW) / dW + 1;
    long nOutputRows = (nInputRows - kH) / dH + 1;

    luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

    input = THCudaTensor_newContiguous(state, input);
    input_data = THCudaTensor_data(state, input);

    THCudaTensor_resize4d(state, output, nbatch, nInputPlane, nOutputRows, nOutputCols);
    THCudaTensor_resize5d(state, indices, 2, nbatch, nInputPlane, nOutputRows, nOutputCols);

    indices_data = THCudaTensor_data(state, indices);
    output_data = THCudaTensor_data(state, output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    // run maxpool kernel
    maxpool <<<blocks, threads>>> (input_data, output_data,
                                   indices_data+nbatch*nInputPlane*nOutputCols*nOutputRows, indices_data,
                                   nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
  }

  // clean
  THCudaTensor_free(state, input);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialMaxsampling.updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static int cunn_SpatialMaxPooling_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  bool atomic = (dW != kW) || (dH != kH);

  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  THCudaTensor *indices = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "indices", "torch.CudaTensor");

  float *indices_data;
  float *gradInput_data;
  float *gradOutput_data;

  if (input->nDimension == 3) {
    long nInputCols = input->size[2];
    long nInputRows = input->size[1];
    long nInputPlane = input->size[0];
    long nOutputCols = gradOutput->size[2];
    long nOutputRows = gradOutput->size[1];

    THCudaTensor_resizeAs(state, gradInput, input);
    THCudaTensor_zero(state, gradInput);

    indices_data = THCudaTensor_data(state, indices);
    gradOutput_data = THCudaTensor_data(state, gradOutput);
    gradInput_data = THCudaTensor_data(state, gradInput);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    if(atomic)
    {
      // run updateGradInput kernel, accumulate gradients atomically
      atomicmaxgradinput <<<blocks, threads>>> (gradInput_data, gradOutput_data,
                                          indices_data+nInputPlane*nOutputCols*nOutputRows, indices_data,
                                          nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    }
    else
    {
      // run updateGradInput kernel
      atomicmaxgradinput <<<blocks, threads>>> (gradInput_data, gradOutput_data,
                                          indices_data+nInputPlane*nOutputCols*nOutputRows, indices_data,
                                          nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    }
  } else {
    long nInputCols = input->size[3];
    long nInputRows = input->size[2];
    long nInputPlane = input->size[1];
    long nbatch = input->size[0];
    long nOutputCols = gradOutput->size[3];
    long nOutputRows = gradOutput->size[2];

    THCudaTensor_resizeAs(state, gradInput, input);
    THCudaTensor_zero(state, gradInput);

    indices_data = THCudaTensor_data(state, indices);
    gradOutput_data = THCudaTensor_data(state, gradOutput);
    gradInput_data = THCudaTensor_data(state, gradInput);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    if(atomic)
    {
      // run updateGradInput kernel, accumulate gradients atomically
      atomicmaxgradinput <<<blocks, threads>>> (gradInput_data, gradOutput_data,
                                          indices_data+nbatch*nInputPlane*nOutputCols*nOutputRows, indices_data,
                                          nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    }
    else
    {
      // run updateGradInput kernel, accumulate gradients atomically
      maxgradinput <<<blocks, threads>>> (gradInput_data, gradOutput_data,
                                          indices_data+nbatch*nInputPlane*nOutputCols*nOutputRows, indices_data,
                                          nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
    }
  }

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialMaxsampling.updateGradInput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static const struct luaL_Reg cunn_SpatialMaxPooling__ [] = {
  {"SpatialMaxPooling_updateOutput", cunn_SpatialMaxPooling_updateOutput},
  {"SpatialMaxPooling_updateGradInput", cunn_SpatialMaxPooling_updateGradInput},
  {NULL, NULL}
};

static void cunn_SpatialMaxPooling_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_SpatialMaxPooling__, "nn");
  lua_pop(L,1);
}

#undef CUDA_MAX_THREADS
