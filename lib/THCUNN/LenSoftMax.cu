#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"

#define LENSOFTMAX_THREADS 128

template <typename T, typename AccumT, typename IndexT>
__global__ void cunn_LenSoftMax_updateOutput_kernel(
  T *output, T *input, int nframe, int dim, IndexT *len)
{
  __shared__ AccumT buffer[LENSOFTMAX_THREADS+1];
  T *input_k  = input  + blockIdx.x*dim + blockIdx.y + blockIdx.z;
  T *output_k = output + blockIdx.x*dim + blockIdx.y + blockIdx.z;

  int i_start = threadIdx.x;
  int i_end = ScalarConvert<IndexT, int>::to(len[blockIdx.x]);
  int i_step = blockDim.x;

  // max?
  buffer[threadIdx.x] = -THCNumerics<AccumT>::max();
  for (int i=i_start; i<i_end; i+=i_step)
  {
    T z = input_k[i];
    AccumT zAcc = ScalarConvert<T, AccumT>::to(z);
    if (buffer[threadIdx.x] < zAcc)
      buffer[threadIdx.x] = zAcc;
  }


  __syncthreads();

  // reduce
  if (threadIdx.x == 0)
  {
    AccumT max_k = -THCNumerics<AccumT>::max();
    for (int i=0; i<blockDim.x; i++)
    {
      if (max_k < buffer[i])
        max_k = buffer[i];
    }
    buffer[LENSOFTMAX_THREADS] = max_k;
  }

  __syncthreads();

  // sum?
  T max_k = ScalarConvert<AccumT, T>::to(buffer[LENSOFTMAX_THREADS]);
  buffer[threadIdx.x] = ScalarConvert<int, AccumT>::to(0);
  for (int i=i_start; i<i_end; i+=i_step) {
    T z = THCNumerics<T>::exp(input_k[i]-max_k);
    buffer[threadIdx.x] += ScalarConvert<T, AccumT>::to(z);
    output_k[i] = z;
  }

  __syncthreads();

  // reduce
  if (threadIdx.x == 0)
  {
    AccumT sum_k = ScalarConvert<int, AccumT>::to(0);
    for (int i=0; i<blockDim.x; i++)
      sum_k += buffer[i];
    buffer[LENSOFTMAX_THREADS] = sum_k;
  }

  __syncthreads();

  // softmax
  T sum_k = ScalarConvert<AccumT, T>::to(buffer[LENSOFTMAX_THREADS]);
  for (int i=i_start; i<i_end; i+=i_step)
    output_k[i] = output_k[i] / sum_k;
}

template <typename T, typename AccumT, typename IndexT>
__global__ void cunn_LenSoftMax_updateGradInput_kernel(
  T *gradInput, T *output, T *gradOutput, int nframe, int dim, IndexT *len)
{
  __shared__ AccumT buffer[LENSOFTMAX_THREADS];
  T *gradInput_k  = gradInput  + blockIdx.x*dim + blockIdx.y + blockIdx.z;
  T *output_k     = output     + blockIdx.x*dim + blockIdx.y + blockIdx.z;
  T *gradOutput_k = gradOutput + blockIdx.x*dim + blockIdx.y + blockIdx.z;

  int i_start = threadIdx.x;
  int i_end = ScalarConvert<IndexT, int>::to(len[blockIdx.x]);
  int i_step = blockDim.x;

  // sum?
  buffer[threadIdx.x] = ScalarConvert<int, AccumT>::to(0);
  for (int i=i_start; i<i_end; i+=i_step)
    buffer[threadIdx.x] += ScalarConvert<T, AccumT>::to(gradOutput_k[i] * output_k[i]);

  __syncthreads();

  // reduce
  if (threadIdx.x == 0)
  {
    AccumT sum_k = ScalarConvert<int, AccumT>::to(0);
    for (int i=0; i<blockDim.x; i++)
      sum_k += buffer[i];
    buffer[0] = sum_k;
  }

  __syncthreads();

  T sum_k = ScalarConvert<AccumT, T>::to(buffer[0]);
  for (int i=i_start; i<i_end; i+=i_step)
    gradInput_k[i] = output_k[i] * (gradOutput_k[i] - sum_k);
}

#include "generic/LenSoftMax.cu"
#include "THCGenerateFloatTypes.h"

#undef LENSOFTMAX_THREADS
