#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"
#include "THCDeviceTensor.cuh"
#include "THCDeviceTensorUtils.cuh"
#include "THCDeviceUtils.cuh"
#include "THCReduceApplyUtils.cuh"
#include <THC/THCApply.cuh>

#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"

template <typename Dtype>
__global__ void SpatialReplicationPadding_updateOutput(
  THCDeviceTensor<Dtype, 4> input,
  THCDeviceTensor<Dtype, 4> output,
  int padT, int padB, int padL, int padR) {

  int outputPointId = threadIdx.x + blockIdx.x * blockDim.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;
  if (outputPointId >= output.getSize(2) * output.getSize(3)) {
    return;
  }
  int outputPointX = outputPointId % output.getSize(3);
  int outputPointY = outputPointId / output.getSize(3);

  int iStartX = max(0, -padL);
  int iStartY = max(0, -padT);
  int oStartX = max(0, padL);
  int oStartY = max(0, padT);

  int inputPointX = min(max(padL, outputPointX), input.getSize(3) + padL - 1) - oStartX + iStartX;
  int inputPointY = min(max(padT, outputPointY), input.getSize(2) + padT - 1) - oStartY + iStartY;

  Dtype valueToCopy = input[batch][plane][inputPointY][inputPointX];
  output[batch][plane][outputPointY][outputPointX] = valueToCopy;
}

template <typename Dtype>
__global__ void SpatialReplicationPadding_updateGradInput(
  THCDeviceTensor<Dtype, 4> gradInput,
  THCDeviceTensor<Dtype, 4> gradOutput,
  int padT, int padB, int padL, int padR) {

  int outputPointId = threadIdx.x + blockIdx.x * blockDim.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;
  if (outputPointId >= gradOutput.getSize(2) * gradOutput.getSize(3)) {
    return;
  }
  int outputPointX = outputPointId % gradOutput.getSize(3);
  int outputPointY = outputPointId / gradOutput.getSize(3);

  int iStartX = max(0, -padL);
  int iStartY = max(0, -padT);
  int oStartX = max(0, padL);
  int oStartY = max(0, padT);

  int inputPointX = min(max(padL, outputPointX), gradInput.getSize(3) + padL - 1) - oStartX + iStartX;
  int inputPointY = min(max(padT, outputPointY), gradInput.getSize(2) + padT - 1) - oStartY + iStartY;

  Dtype valueToCopy = gradOutput[batch][plane][outputPointY][outputPointX];
  atomicAdd(&gradInput[batch][plane][inputPointY][inputPointX], valueToCopy);
}


#include "generic/SpatialReplicationPadding.cu"
#include "THCGenerateFloatTypes.h"
