#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>

struct kl_functor
{
  __host__ __device__ float operator()(const float& x, const float& y) const
  {
      return y > 0 ? y * (log(y) - x) : 0;
  }
};

void THNN_CudaDistKLDivCriterion_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *target, THCudaTensor *output, bool sizeAverage)
{
  THCUNN_assertSameGPU(state, 2, input, target);

  THArgCheck(THCudaTensor_nElement(state, input) == THCudaTensor_nElement(state, target), 2,
             "input and target need to have the same number of elements");

  float sum;

  ptrdiff_t size = THCudaTensor_nElement(state, input);

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  sum = thrust::inner_product(input_data, input_data+size, target_data, (float) 0, thrust::plus<float>(), kl_functor());

  if (sizeAverage)
    sum /= size;

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);

  THCudaTensor_set1d(state, output, 0, sum);
}

struct kl_updateGradInput_functor
{
  const float norm;

  kl_updateGradInput_functor(float norm_)
    : norm(norm_)
  {}

  __host__ __device__ float operator()(const float& x, const float& y) const
  {
      return y > 0 ? norm * (-y) : 0;
  }
};

void THNN_CudaDistKLDivCriterion_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *target, THCudaTensor *gradInput, bool sizeAverage)
{
  THCUNN_assertSameGPU(state, 3, input, target, gradInput);

  THArgCheck(THCudaTensor_nElement(state, input) == THCudaTensor_nElement(state, target), 2,
             "input and target need to have the same number of elements");

  ptrdiff_t size = THCudaTensor_nElement(state, input);
  float norm = (sizeAverage ? 1./size : 1.);

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

  THCudaTensor_resizeAs(state, gradInput, input);

  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  thrust::device_ptr<float> gradInput_data(THCudaTensor_data(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data, kl_updateGradInput_functor(norm));

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);
}
