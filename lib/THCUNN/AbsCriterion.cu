#include "hip/hip_runtime.h"
#include "THCUNN.h"

#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>

struct abs_functor
{
  abs_functor() {}

  __host__ __device__ float operator()(const float& x, const float& y) const
    {
      float z = x-y;
      return z >= 0 ? z : -z;
    }
};

void THNN_CudaAbsCriterion_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *target, float *output, bool sizeAverage)
{
  THAssert(THCudaTensor_checkGPU(state, 2, input, target));

  long size = THCudaTensor_nElement(state, input);

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  float sum = thrust::inner_product(input_data, input_data+size, target_data, (float) 0, thrust::plus<float>(), abs_functor());

  if (sizeAverage)
    sum /= size;

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);

  *output = sum;
}


struct abs_updateGradInput_functor
{
  const float norm;

  abs_updateGradInput_functor(float norm_) : norm(norm_) {}

  __host__ __device__ float operator()(const float& x, const float& y) const
    {
      return (x - y) >= 0 ? norm : -norm;
    }
};

void THNN_CudaAbsCriterion_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *target, THCudaTensor *gradInput, bool sizeAverage)
{
  THAssert(THCudaTensor_checkGPU(state, 3, input, target, gradInput));

  long size = THCudaTensor_nElement(state, input);
  float norm = (sizeAverage ? 1./size : 1.);

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

  THCudaTensor_resizeAs(state, gradInput, input);

  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  thrust::device_ptr<float> gradInput_data(THCudaTensor_data(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data, abs_updateGradInput_functor(norm));

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);
}
