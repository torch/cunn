#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

#include <stdio.h>
#include <assert.h>

static const int NTHREADS = 32;

__global__ void cunn_WeightedClassNLLCriterion_updateOutput_kernel1(float *output,
                                                           float *total_weight,
                                                           float *input,
                                                           float *target,
                                                           float *weights,
                                                           int size_average,
                                                           int n_classes) {
  assert(threadIdx.x == 0 && threadIdx.y == 0 && threadIdx.z == 0);

  // TODO: T4951791 Reuse code between updateOutput_kernel1 and
  // updateOutput_kernel.

  int t = (int)*target - 1;
  assert(t >= 0 && t < n_classes);
  float cur_weight = weights ? weights[0] : 1.0f;
  *output = -cur_weight * input[t];
  *total_weight = cur_weight;
  if (size_average && *total_weight > 0) {
    *output /= *total_weight;
  }
}

__global__ void cunn_WeightedClassNLLCriterion_updateOutput_kernel(float *output,
                                                           float *total_weight,
                                                           float *input,
                                                           float *target,
                                                           float *weights,
                                                           int size_average,
                                                           int nframe,
                                                           int ndim,
                                                           int n_classes) {
  __shared__ float shInputs[NTHREADS], acc_weight[NTHREADS];
  int i, t;
  float cur_weight;

  shInputs[threadIdx.x] = 0.0f;
  acc_weight[threadIdx.x] = 0.0f;
  for (i = threadIdx.x; i < nframe; i += NTHREADS) {
      t = target[i] - 1;
      assert(t >= 0 && t < n_classes);
      cur_weight = weights ? weights[i] : 1.0f;
      shInputs[threadIdx.x] -= input[i * ndim + t] * cur_weight;
      acc_weight[threadIdx.x] += cur_weight;
  }
  __syncthreads();

  // TODO: T4951791 Reuse code between updateOutput_kernel1 and
  // updateOutput_kernel

  if (threadIdx.x == 0) {
    *output = *total_weight = 0;
    for (i = 0; i < NTHREADS; ++i){
      *output += shInputs[i];
      *total_weight += acc_weight[i];
    }
    if (size_average && *total_weight > 0) {
      *output /= *total_weight;
    }
  }
}

__global__ void cunn_WeightedClassNLLCriterion_updateGradInput_kernel1(
  float* gradInput,
  float* weights,
  float* target,
  float* total_weight,
  int size_average,
  int n_classes)
{
  if (*total_weight <= 0) {
    return;
  }
  float norm = size_average ? (1.0f / *total_weight) : 1.0f;
  int t = (int)*target - 1;
  assert(t >= 0 && t < n_classes);
  gradInput[t] = -(weights ? weights[0] : 1.0f) * norm;
}

__global__ void cunn_WeightedClassNLLCriterion_updateGradInput_kernel(
  float *gradInput,
  float *target,
  float *weights,
  float *total_weight,
  int size_average,
  int nframe,
  int ndim,
  int n_classes)
{
  if (*total_weight <= 0) {
    return;
  }
  int i, t;
  float norm = size_average ? (1.0f / *total_weight) : 1.0f;

  for (i = threadIdx.x; i < nframe; i += NTHREADS) {
    t = (int)target[i] - 1;
    assert(t >= 0 && t < n_classes);
    gradInput[i * ndim + t] = -(weights ? weights[i] : 1.0f) * norm;
  }
}

void THNN_CudaWeightedClassNLLCriterion_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *target, THCudaTensor *output, bool sizeAverage, THCudaTensor *weights, THCudaTensor *total_weight) {
  if (THCudaTensor_nDimension(state, target) > 1) {
    THError("multi-target not supported");
  }

  int n_dims = THCudaTensor_nDimension(state, input);
  int n_classes = THCudaTensor_size(state, input, n_dims - 1);

  if (weights) {
    THCUNN_assertSameGPU(
      state, 5, input, target, weights, output, total_weight
    );
  } else {
    THCUNN_assertSameGPU(
      state, 4, input, target, output, total_weight
    );
  }

  if (THCudaTensor_nDimension(state, input) > 2) {
    THArgCheck(0, 2, "vector or matrix expected");
  }

  input = THCudaTensor_newContiguous(state, input);
  weights = weights ? THCudaTensor_newContiguous(state, weights) : NULL;
  target = THCudaTensor_newContiguous(state, target);

  float *input_data = THCudaTensor_data(state, input);
  float *weights_data = weights ? THCudaTensor_data(state, weights) : NULL;
  float *target_data = THCudaTensor_data(state, target);
  float *output_data = THCudaTensor_data(state, output);
  float *total_weight_data = THCudaTensor_data(state, total_weight);

  if (THCudaTensor_nDimension(state, input) == 1) {
    cunn_WeightedClassNLLCriterion_updateOutput_kernel1
      <<<1, 1, 0, THCState_getCurrentStream(state)>>>(
        output_data,
        total_weight_data,
        input_data,
        target_data,
        weights_data,
        sizeAverage,
        n_classes
    );

  } else if (THCudaTensor_nDimension(state, input) == 2) {
    cunn_WeightedClassNLLCriterion_updateOutput_kernel
      <<<1, NTHREADS, 0, THCState_getCurrentStream(state)>>>(
        output_data,
        total_weight_data,
        input_data,
        target_data,
        weights_data,
        sizeAverage,
        THCudaTensor_size(state, input, 0),
        THCudaTensor_size(state, input, 1),
        n_classes
    );
  }

  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess) {
    THError(hipGetErrorString(errcode));
  }
  if (weights) {
    THCudaTensor_free(state, weights);
  }
  THCudaTensor_free(state, target);
  THCudaTensor_free(state, input);
}

void THNN_CudaWeightedClassNLLCriterion_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *target, THCudaTensor *gradInput, bool sizeAverage, THCudaTensor *weights, THCudaTensor *total_weight) {
  if (THCudaTensor_nDimension(state, target) > 1) {
    THError("multi-target not supported");
  }

  int n_dims = THCudaTensor_nDimension(state, input);
  int n_classes = THCudaTensor_size(state, input, n_dims - 1);

  THArgCheck(THCudaTensor_isContiguous(state, gradInput), 4, "gradInput must be contiguous");

  if (weights) {
    THCUNN_assertSameGPU(
      state, 5, weights, input, target, gradInput, total_weight
    );
  }
  else {
    THCUNN_assertSameGPU(
      state, 4, input, target, gradInput, total_weight
    );
  }

  if (THCudaTensor_nDimension(state, input) > 2) {
    THArgCheck(0, 2, "vector or matrix expected");
  }

  weights = weights ? THCudaTensor_newContiguous(state, weights) : NULL;
  target = THCudaTensor_newContiguous(state, target);

  float *weights_data = weights ? THCudaTensor_data(state, weights) : NULL;
  float *gradInput_data = THCudaTensor_data(state, gradInput);
  float *target_data = THCudaTensor_data(state, target);
  float *total_weight_data = THCudaTensor_data(state, total_weight);

  if (THCudaTensor_nDimension(state, input) == 1) {
    cunn_WeightedClassNLLCriterion_updateGradInput_kernel1
      <<<1, 1, 0, THCState_getCurrentStream(state)>>>(
        gradInput_data,
        weights_data,
        target_data,
        total_weight_data,
        sizeAverage,
        n_classes
    );
  } else {
    cunn_WeightedClassNLLCriterion_updateGradInput_kernel
      <<<1, NTHREADS, 0, THCState_getCurrentStream(state)>>>(
        gradInput_data,
        target_data,
        weights_data,
        total_weight_data,
        sizeAverage,
        THCudaTensor_size(state, input, 0),
        THCudaTensor_size(state, input, 1),
        n_classes
    );
  }
  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess) {
    THError(hipGetErrorString(errcode));
  }
  if (weights) {
    THCudaTensor_free(state, weights);
  }
  THCudaTensor_free(state, target);
}
