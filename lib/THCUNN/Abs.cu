#include "hip/hip_runtime.h"
#include "THCUNN.h"

struct absupdateOutput_functor
{
  __device__ void operator()(float* output, const float* input) const
  {
    *output = abs(*input);
  }
};

void THNN_CudaAbs_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output)
{
  THAssert(THCudaTensor_checkGPU(state, 2, input, output));
  THCudaTensor_resizeAs(state, output, input);
  THCudaTensor_pointwiseApply2(state, output, input, absupdateOutput_functor());
}

struct absupdateGradInput_functor
{
  __device__ void operator()(float* gradInput, const float* input, const float* gradOutput) const
  {
    *gradInput = *input < 0 ? - *gradOutput : *gradOutput;
  }
};

void THNN_CudaAbs_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *gradOutput, THCudaTensor *gradInput)
{
  THAssert(THCudaTensor_checkGPU(state, 3, input, gradOutput, gradInput));
  THCudaTensor_resizeAs(state, gradInput, input);
  THCudaTensor_pointwiseApply3(state, gradInput, input, gradOutput, absupdateGradInput_functor());
}
