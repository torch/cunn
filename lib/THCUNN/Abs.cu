#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include <THC/THCApply.cuh>

template <typename T>
struct absupdateOutput_functor
{
  __device__ void operator()(T* output, const T* input) const
  {
    *output = abs(*input);
  }
};

template <typename T>
struct absupdateGradInput_functor
{
  __device__ void operator()(T* gradInput, const T* input, const T* gradOutput) const
  {
    *gradInput = *input < 0 ? - *gradOutput : *gradOutput;
  }
};

#include "generic/Abs.cu"
#include "THCGenerateFloatTypes.h"
