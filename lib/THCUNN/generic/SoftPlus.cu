
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/SoftPlus.cu"
#else

#include "../common.h"

void THNN_(SoftPlus_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *output,
           real beta,
           real threshold)
{
  THCUNN_assertSameGPU_generic(state, 2, input, output);
  THCTensor_(resizeAs)(state, output, input);
  THC_pointwiseApply2(state, output, input, softPlusupdateOutput_functor<real>(threshold, beta));
}

void THNN_(SoftPlus_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *gradOutput,
           THCTensor *gradInput,
           THCTensor *output,
           real beta,
           real threshold)
{
  THCUNN_assertSameGPU_generic(state, 4, input, output, gradOutput, gradInput);
  THCTensor_(resizeAs)(state, gradInput, output);
  THC_pointwiseApply3(state, gradInput, output, gradOutput, softPlusupdateGradInput_functor<real>(threshold, beta));
}

#endif
