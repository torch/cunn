
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "generic/MSECriterion.cu"
#else

void THNN_(MSECriterion_updateOutput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *output,
           bool sizeAverage)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_check_dim_size(state, output, 1, 0, 1);
  THCUNN_assertSameGPU(state, 2, input, target);

  ptrdiff_t size = THCTensor_(nElement)(state, input);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  THCThrustAllocator thrustAlloc(state);
  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  accreal sum = thrust::inner_product(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
    input_data, input_data+size, target_data, (accreal) 0,
    thrust::plus<accreal>(), mse_functor<real, accreal>());

  if (sizeAverage)
    sum /= size;

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);

  THCTensor_(set1d)(state, output, 0, ScalarConvert<accreal, real>::to(sum));
}

void THNN_(MSECriterion_updateGradInput)(
           THCState *state,
           THCTensor *input,
           THCTensor *target,
           THCTensor *gradInput,
           bool sizeAverage)
{
  THCUNN_check_nElement(state, input, target);
  THCUNN_assertSameGPU(state, 3, input, target, gradInput);

  ptrdiff_t size = THCTensor_(nElement)(state, input);
  accreal norm = sizeAverage ? (accreal)(2)/size : (accreal)(2);

  input = THCTensor_(newContiguous)(state, input);
  target = THCTensor_(newContiguous)(state, target);

  THCTensor_(resizeAs)(state, gradInput, input);

  THCThrustAllocator thrustAlloc(state);
  thrust::device_ptr<real> input_data(THCTensor_(data)(state, input));
  thrust::device_ptr<real> target_data(THCTensor_(data)(state, target));
  thrust::device_ptr<real> gradInput_data(THCTensor_(data)(state, gradInput));

  thrust::transform(
#if CUDA_VERSION >= 7000
    thrust::cuda::par(thrustAlloc).on(THCState_getCurrentStream(state)),
#endif
    input_data, input_data+size, target_data, gradInput_data,
    mse_updateGradInput_functor<real, accreal>(norm));

  THCTensor_(free)(state, input);
  THCTensor_(free)(state, target);
}

#endif
