#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include <THC/THCApply.cuh>

template <typename T>
struct hardtanhupdateOutput_functor
{
  const T max_val_;
  const T min_val_;

  hardtanhupdateOutput_functor(T min_val, T max_val)
    : min_val_(min_val)
    , max_val_(max_val)
  {}

  __device__ void operator()(T *output, const T *input) const
  {
    if (*input < min_val_)
      *output = min_val_;
    else if (*input <= max_val_)
      *output = *input;
    else
      *output = max_val_;
  }

  __device__ void operator()(T *input) const
  {
    if (*input < min_val_)
      *input = min_val_;
    else if (*input > max_val_)
      *input = max_val_;
  }
};

template <typename T>
struct hardtanhupdateGradInput_functor
{
  const T max_val_;
  const T min_val_;

  hardtanhupdateGradInput_functor(T min_val, T max_val)
    : min_val_(min_val)
    , max_val_(max_val)
  {}

  __device__ void operator()(T *gradInput, const T *input, const T *gradOutput) const
  {
    if (*input < min_val_ || *input > max_val_)
      *gradInput = ScalarConvert<int, T>::to(0);
    else
      *gradInput = *gradOutput;
  }

  __device__ void operator()(T *gradInput, const T *input) const
  {
    if (*input <= min_val_ || *input >= max_val_)
      *gradInput = ScalarConvert<int, T>::to(0);
  }
};

#include "generic/HardTanh.cu"
#include "THCGenerateFloatTypes.h"
