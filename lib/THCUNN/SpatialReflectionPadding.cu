#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"
#include "THCDeviceTensor.cuh"
#include "THCDeviceTensorUtils.cuh"
#include "THCDeviceUtils.cuh"
#include "THCReduceApplyUtils.cuh"
#include <THC/THCApply.cuh>

#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"

template<typename Dtype>
__global__ void SpatialReflectionPadding_updateOutput(
  THCDeviceTensor<Dtype, 4> input,
  THCDeviceTensor<Dtype, 4> output,
  int padT, int padB, int padL, int padR) {

  int outputPointId = threadIdx.x + blockIdx.x * blockDim.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;
  if (outputPointId >= output.getSize(2) * output.getSize(3)) {
    return;
  }
  int outputPointX = outputPointId % output.getSize(3);
  int outputPointY = outputPointId / output.getSize(3);

  int iStartX = max(0, -padL);
  int iStartY = max(0, -padT);
  int oStartX = max(0, padL);
  int oStartY = max(0, padT);

  int inputPointX = abs(outputPointX - padL)
                  - abs(outputPointX - (input.getSize(3) + padL - 1))
                  - outputPointX
                  + 2 * padL + input.getSize(3) - 1
                  - oStartX + iStartX;

  int inputPointY = abs(outputPointY - padT)
                  - abs(outputPointY - (input.getSize(2) + padT - 1))
                  - outputPointY
                  + 2 * padT + input.getSize(2) - 1
                  - oStartY + iStartY;

  Dtype valueToCopy = input[batch][plane][inputPointY][inputPointX];
  output[batch][plane][outputPointY][outputPointX] = valueToCopy;
}

template <typename Dtype>
__global__ void SpatialReflectionPadding_updateGradInput(
  THCDeviceTensor<Dtype, 4> gradInput,
  THCDeviceTensor<Dtype, 4> gradOutput,
  int padT, int padB, int padL, int padR) {

  int outputPointId = threadIdx.x + blockIdx.x * blockDim.x;
  int plane = blockIdx.y;
  int batch = blockIdx.z;
  if (outputPointId >= gradOutput.getSize(2) * gradOutput.getSize(3)) {
    return;
  }
  int outputPointX = outputPointId % gradOutput.getSize(3);
  int outputPointY = outputPointId / gradOutput.getSize(3);

  int iStartX = max(0, -padL);
  int iStartY = max(0, -padT);
  int oStartX = max(0, padL);
  int oStartY = max(0, padT);

  int inputPointX = abs(outputPointX - padL)
                  - abs(outputPointX - (gradInput.getSize(3) + padL - 1))
                  - outputPointX
                  + 2 * padL + gradInput.getSize(3) - 1
                  - oStartX + iStartX;

  int inputPointY = abs(outputPointY - padT)
                  - abs(outputPointY - (gradInput.getSize(2) + padT - 1))
                  - outputPointY
                  + 2 * padT + gradInput.getSize(2) - 1
                  - oStartY + iStartY;

  Dtype valueToCopy = gradOutput[batch][plane][outputPointY][outputPointX];
  atomicAdd(&gradInput[batch][plane][inputPointY][inputPointX], valueToCopy);
}

#include "generic/SpatialReflectionPadding.cu"
#include "THCGenerateFloatTypes.h"
