#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCAtomics.cuh"
#include "common.h"
#include <THC/THCApply.cuh>

#include <thrust/functional.h>

template <typename T, typename AccumT>
__global__ void cunn_SpatialClassNLLCriterion_updateOutput_kernel(
          T *output,
          T *total_weight,
          T *input,
          THCIndex_t *target,
          T *weights,
          int size_average,
          int batch_size,
          int n_classes,
          int map_nelem,
          int blocks_per_sample)
{
  __shared__ AccumT partial_sums[CUDA_NUM_THREADS];

  int i, t;
  T cur_weight;
  AccumT input_sum = 0;
  AccumT acc_weight = 0;

  int sample = blockIdx.x / blocks_per_sample;
  int toffset = sample * map_nelem;
  int ioffset = sample * map_nelem * n_classes;
  int step = blockDim.x * blocks_per_sample;
  for (i = (blockIdx.x % blocks_per_sample) * blockDim.x + threadIdx.x;
       i < map_nelem;
       i += step) {
    t = target[toffset + i] - TH_INDEX_BASE;
    assert(t >= 0 && t < n_classes);
    cur_weight = weights ? weights[t] : ScalarConvert<int, T>::to(1);
    input_sum -= input[ioffset + i + map_nelem * t] * cur_weight;
    acc_weight += cur_weight;
  }

  __syncthreads();

  input_sum = reduceBlock(partial_sums, blockDim.x, input_sum, thrust::plus<AccumT>(), AccumT(0));
  acc_weight = reduceBlock(partial_sums, blockDim.x, acc_weight, thrust::plus<AccumT>(), AccumT(0));

  if (threadIdx.x == 0) {
    atomicAdd(total_weight, ScalarConvert<AccumT, T>::to(acc_weight));
    if (size_average && acc_weight > 0)
      atomicAdd(output, ScalarConvert<AccumT, T>::to(input_sum / acc_weight / gridDim.x));
    else
      atomicAdd(output, ScalarConvert<AccumT, T>::to(input_sum));
  }
}

template<typename T>
__global__ void cunn_SpatialClassNLLCriterion_updateGradInput_kernel(
          T *gradInput,
          THCIndex_t *target,
          T *weights,
          T *total_weight,
          int size_average,
          int batch_size,
          int n_classes,
          int map_nelem,
          int blocks_per_sample)
{
  if (*total_weight <= 0)
    return;

  int i, t;
  T norm = size_average ? (ScalarConvert<int, T>::to(1) / *total_weight) : ScalarConvert<int, T>::to(1);

  int sample = blockIdx.x / blocks_per_sample;
  int step = blockDim.x * blocks_per_sample;
  int toffset = sample * map_nelem;
  int ioffset = sample * map_nelem * n_classes;
  for (i = (blockIdx.x % blocks_per_sample) * blockDim.x + threadIdx.x;
       i < map_nelem;
       i += step) {
    t = (int)target[toffset + i] - TH_INDEX_BASE;
    assert(t >= 0 && t < n_classes);
    gradInput[ioffset + i + map_nelem * t] = -(weights ? weights[t] : ScalarConvert<int, T>::to(1)) * norm;
  }
}

#include "generic/SpatialClassNLLCriterion.cu"
#include "THCGenerateFloatTypes.h"
