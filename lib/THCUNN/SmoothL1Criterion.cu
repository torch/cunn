#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"
#include "THCHalf.h"
#include "THCHalfAutoNumerics.cuh"
#include "THCThrustAllocator.cuh"

#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>
#if CUDA_VERSION >= 7000
#include <thrust/system/cuda/execution_policy.h>
#endif

template <typename Dtype, typename Acctype>
struct smoothl1_functor
{
  smoothl1_functor() {}

  __host__ __device__ Acctype operator()(const Dtype &x, const Dtype &y) const
  {
    Acctype z = ScalarConvert<Dtype, Acctype>::to(THCNumerics<Dtype>::abs(x-y));
    return z < Acctype(1) ? 0.5f*z*z : z - 0.5f;
  }
};

template <typename Dtype>
struct smoothl1_updateGradInput_functor
{
  const Dtype norm;

  smoothl1_updateGradInput_functor(Dtype norm_)
    : norm(norm_)
  {}

  __host__ __device__ Dtype operator()(const Dtype &x, const Dtype &y) const
  {
    Dtype z = x - y;
    if (z < ScalarConvert<int, Dtype>::to(-1))
      return -norm;
    else if (z > ScalarConvert<int, Dtype>::to(1))
      return norm;
    else
      return norm * z;
  }
};

#include "generic/SmoothL1Criterion.cu"
#include "THCGenerateFloatTypes.h"
