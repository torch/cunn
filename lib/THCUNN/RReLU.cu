#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

// copied from cutorch/lib/THC/THCTensorRandom.cu
#define MAX_NUM_BLOCKS 64
#define BLOCK_SIZE 256
#define NUM_BLOCKS(n) min((int)THCCeilDiv(n, (ptrdiff_t) BLOCK_SIZE), MAX_NUM_BLOCKS)

__global__ void rreluUpdateOutputTrain(ptrdiff_t n, hiprandStateMtgp32 *state,
  float *input, float* noise, float *output, double a, double b)
{
  CUDA_KERNEL_LOOP(i, n)
  {
    if (input[i] <= 0)
    {
      float r = hiprand_uniform(&state[blockIdx.x]);
      r = r * (b-a) + a;
      output[i] = input[i] * r;
      noise[i] = r;
    }
    else
    {
      output[i] = input[i];
      noise[i] = 1;
    }
  }
}

struct RReLUUpdateOutputEval_functor
{
  const float negSlope_;

  RReLUUpdateOutputEval_functor(float negSlope)
    : negSlope_(negSlope)
  {}

  __device__ __forceinline__ void operator()(float *out, float *in)
  {
    const float x = *in;
    const float r = x <= 0 ? negSlope_ : 1;
    *out = x * r;
  }
};

struct RReLUUpdateOutputEvalIP_functor
{
  const float negSlope_;

  RReLUUpdateOutputEvalIP_functor(float negSlope)
    : negSlope_(negSlope)
  {}

  __device__ __forceinline__ void operator()(float *x)
  {
    if (*x <= 0)
    {
      *x = *x * negSlope_;
    }
  }
};

void THNN_CudaRReLU_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output,
  THCudaTensor *noise, double lower, double upper, bool train, bool inplace, void *generator)
{
  THCUNN_assertSameGPU(state, 3, input, output, noise);
  struct hiprandStateMtgp32* gen_states = THCRandom_generatorStates(state);

  if (train)
  {
    input = THCudaTensor_newContiguous(state, input);
    THCudaTensor_resizeAs(state, noise, input);
    float *input_data = THCudaTensor_data(state, input);
    float *noise_data = THCudaTensor_data(state, noise);
    ptrdiff_t n = THCudaTensor_nElement(state, input);
    if (inplace)
    {
      rreluUpdateOutputTrain<<<NUM_BLOCKS(n), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
        n, gen_states, input_data, noise_data, input_data, lower, upper);
      THCudaTensor_set(state, output, input);
    }
    else
    {
      THCudaTensor_resizeAs(state, output, input);
      float *output_data = THCudaTensor_data(state, output);
      rreluUpdateOutputTrain<<<NUM_BLOCKS(n), BLOCK_SIZE, 0, THCState_getCurrentStream(state)>>>(
        n, gen_states, input_data, noise_data, output_data, lower, upper);
    }
    THCudaCheck(hipGetLastError());
    THCudaTensor_free(state, input);
  }
  else
  {
    const double negSlope = (lower + upper) / 2;
    if (inplace)
    {
      THC_pointwiseApply1(state, input, RReLUUpdateOutputEvalIP_functor(negSlope));
      THCudaTensor_set(state, output, input);
    }
    else
    {
      THCudaTensor_resizeAs(state, output, input);
      THC_pointwiseApply2(state, output, input, RReLUUpdateOutputEval_functor(negSlope));
    }
  }
}

struct RReLUupdateGradInputEval_functor
{
  const float negSlope_;

  RReLUupdateGradInputEval_functor(float negSlope)
    : negSlope_(negSlope)
  {}

  __device__ __forceinline__ void operator()(float *gradIn, float *gradOut, float *in)
  {
    *gradIn = (*in) <= 0 ? (*gradOut) * negSlope_ : (*gradOut);
  }
};

struct RReLUupdateGradInputEvalIP_functor
{
  const float negSlope_;

  RReLUupdateGradInputEvalIP_functor(float negSlope)
    : negSlope_(negSlope)
  {}

  __device__ __forceinline__ void operator()(float *gradOut, float *in)
  {
    if (*in <= 0)
    {
      *gradOut = (*gradOut) * negSlope_;
    }
  }
};

void THNN_CudaRReLU_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *gradOutput,
  THCudaTensor *gradInput, THCudaTensor *noise, double lower, double upper, bool train, bool inplace)
{
  THCUNN_assertSameGPU(state, 4, input, gradOutput, gradInput, noise);

  gradOutput = THCudaTensor_newContiguous(state, gradOutput);

  if (train && upper - lower > 1E-6)    // e.g. if upper == lower, RReLU behaves like LeakyReLU
  {
    // multiply the gradient by the noise tensor
    if (inplace)
    {
      THCudaTensor_cmul(state, gradOutput, gradOutput, noise);
      THCudaTensor_set(state, gradInput, gradOutput);
    }
    else
    {
      THCudaTensor_resizeAs(state, gradInput, input);
      THCudaTensor_cmul(state, gradInput, gradOutput, noise);
    }
  }
  else
  {
    // use constant factor for negative input values
    const double negSlope = (lower + upper) / 2;
    if (inplace)
    {
      THC_pointwiseApply2(state, gradOutput, input, RReLUupdateGradInputEvalIP_functor(negSlope));
      THCudaTensor_set(state, gradInput, gradOutput);
    }
    else
    {
      THCudaTensor_resizeAs(state, gradInput, input);
      THC_pointwiseApply3(state, gradInput, gradOutput, input, RReLUupdateGradInputEval_functor(negSlope));
    }
  }

  THCudaTensor_free(state, gradOutput);
}
