#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/transform_reduce.h>
#if CUDA_VERSION >= 7000
#include <thrust/system/cuda/execution_policy.h>
#endif
#include <thrust/unique.h>

#ifndef DIVUP
#define DIVUP(x, y) (((x) + (y) - 1) / (y))
#endif

const int WARP_SIZE = 32;

__device__ __forceinline__ bool warpHasCollision(int val)
{
  // Compare our value to the values stored in the next 16 lanes,
  // wrapping around at 32. If any pair of values is the same than
  // there is a collision in the warp.
  bool dup = 0;
  const int laneId = threadIdx.x % 32;

#if __CUDA_ARCH__ >= 300

  #pragma unroll
  for (int i = 1; i <= 16; i++)
  {
    dup |= (__shfl(val, (laneId + i) % 32) == val);
  }

#else

  volatile __shared__ int values[128];
  values[threadIdx.x] = val;
  const int offset = threadIdx.x - laneId;

  #pragma unroll
  for (int i = 1; i <= 16; i++)
  {
    dup |= (values[offset + ((laneId + i) % 32)] == val);
  }

#endif

  return __any(dup) != 0;
}

__global__ void cunn_LookupTable_accGradParametersKernelByFeature(
  long *input, float *gradOutput, float *gradWeight, float scale, ptrdiff_t numel,
  long stride, int paddingValue) {

  const int featureDim = blockIdx.x * 4 + threadIdx.x / 32;
  if (featureDim >= stride) {
    return;
  }

  // The strategy here is that each warp handles a single feature
  // dimension.
  // Within that feature dimension, points in the [batch][element]
  // dimension can overlap, and we need to determine if threads want
  // to add to the gradient in a colliding manner.
  // Typically one would use floating-point atomicAdd() to resolve
  // these collisions, but that is non-deterministic if there are
  // collisions. Non-determinism for this code is really bad,
  // especially in RNNs, and is prone to snowballing error.
  // In order to get a deterministic order of execution, we handle
  // non-colliding updates separately from colliding ones. Colliding
  // updates are serialized in their order of execution by using the
  // warp-wide collision detector `warpHasCollision`.
  const int laneId = threadIdx.x % 32;
  for (ptrdiff_t i = laneId; i < numel; i += WARP_SIZE) {
    const int weightIndex = (int) (input[i] - TH_INDEX_BASE);
    if (weightIndex == paddingValue - TH_INDEX_BASE) {
      continue;
    }

    float update = gradOutput[i*stride + featureDim] * scale;

    // Check for collision
    if (warpHasCollision(weightIndex)) {
      // Run all lanes sequentially; warp divergence
      for (int i = 0; i < WARP_SIZE; ++i) {
        if (laneId == i) {
          gradWeight[weightIndex*stride + featureDim] += update;
        }
      }
    } else {
      // No collision; warp coherence
      gradWeight[weightIndex*stride + featureDim] += update;
    }
  }
}

__global__ void cunn_LookupTable_accGradParametersKernel(
  long *input, long *indices, float *gradOutput, float *gradWeight,
  long *count, float defaultScale, ptrdiff_t numel, long stride, int paddingValue) {

  int idx = blockIdx.x * 4 + threadIdx.y;

  // Each warp is responsible for an input into the LookupTable.
  // If the preceeding input has the same as this input, then the warp
  // exits immediately. The warp also processes subsequent inputs with the
  // same value.
  //
  // Input Warp
  // 1     <warp 1>
  // 1     <warp 1> (<warp 2> exits without doing any work)
  // 5     <warp 3>
  // 8     <warp 4>

  // Number of values proceessed by each thread (grain size)
  const int SZ = 4;

  if (idx < numel
      && (idx == 0 || input[idx] != input[idx - 1])
      && input[idx] != paddingValue) {
    do {
      const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
      const int weightRow = ((int) input[idx] - TH_INDEX_BASE) * stride;
      const int gradOutputRow = ((int) indices[idx] - TH_INDEX_BASE) * stride;
      const float scale = count ? defaultScale / count[idx] : defaultScale;

      float gradient[SZ];
      float weight[SZ];

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++)
      {
        int featureDim = startFeature + ii * WARP_SIZE;
        if (featureDim < stride)
        {
          gradient[ii] = gradOutput[gradOutputRow + featureDim];
          weight[ii] = gradWeight[weightRow + featureDim];
        }
      }

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++)
      {
        weight[ii] += gradient[ii] * scale;
      }

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++)
      {
        int featureDim = startFeature + ii * WARP_SIZE;
        if (featureDim < stride)
        {
          gradWeight[weightRow + featureDim] = weight[ii];
        }
      }

      idx++;
    } while (idx < numel && input[idx] == input[idx - 1]);
  }
}

void THNN_CudaLookupTable_accGradParameters(
  THCState *state,
  THIndexTensor *input,
  THCudaTensor *gradOutput,
  THCudaTensor *gradWeight,
  THIndexTensor *count,
  THIndexTensor *sorted,
  THIndexTensor *indices,
  bool scaleGradByFreq,
  int paddingValue,
  float scale)
{
  THCUNN_assertSameGPU(state, 5, input, gradOutput, gradWeight, sorted, indices);
  if (!(THIndexTensor_(isContiguous)(state, input) &&
        THCudaTensor_isContiguous(state, gradOutput) &&
        THCudaTensor_isContiguous(state, gradWeight)))
  {
    THError("Tensors must be contiguous");
  }

  int nDim = THIndexTensor_(nDimension)(state, input);
  if (nDim != 1 && nDim != 2)
    THError("input must be a vector or matrix");

  ptrdiff_t numel = THIndexTensor_(nElement)(state, input);
  long stride = gradWeight->stride[0];

  hipStream_t stream = THCState_getCurrentStream(state);

  if (numel <= 768 && !scaleGradByFreq) {
    cunn_LookupTable_accGradParametersKernelByFeature<<<DIVUP(stride,4), 128, 0, stream>>>(
      THIndexTensor_(data)(state, input),
      THCudaTensor_data(state, gradOutput),
      THCudaTensor_data(state, gradWeight),
      scale,
      numel,
      stride,
      paddingValue);
    THCudaCheck(hipGetLastError());
    return;
  }

  THLongStorage *inputSize = THIndexTensor_(newSizeOf)(state, input);
  THIndexTensor_(resize)(state, sorted, inputSize, NULL);
  THIndexTensor_(resize)(state, indices, inputSize, NULL);
  THLongStorage_free(inputSize);

  // Sort the inputs into sorted with the corresponding indices
  THIndexTensor_(sort)(state, sorted, indices, input, 0, 0);

  long *sorted_data = THIndexTensor_(data)(state, sorted);
  long  *indices_data = THIndexTensor_(data)(state, indices);
  long *count_data = NULL;

  if (scaleGradByFreq)
  {
    THIndexTensor_(resizeAs)(state, count, input);
    count_data = THIndexTensor_(data)(state, count);

    thrust::device_ptr<long> sorted_ptr(sorted_data);
    thrust::device_ptr<long> count_ptr(count_data);

    // Compute an increasing sequence per unique item in sorted:
    // sorted: 2 5 5 5 7 7 8 9 9
    //  count: 1 1 2 3 1 2 1 1 2
    thrust::inclusive_scan_by_key(
#if CUDA_VERSION >= 7000
      thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
      sorted_ptr,
      sorted_ptr + numel,
      thrust::make_constant_iterator(1),
      count_ptr
    );

    // Take the maximum of each count per unique key in reverse:
    // sorted: 2 5 5 5 7 7 8 9 9
    //  count: 1 3 3 3 2 2 1 2 2
    thrust::inclusive_scan_by_key(
#if CUDA_VERSION >= 7000
      thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
      thrust::make_reverse_iterator(sorted_ptr + numel),
      thrust::make_reverse_iterator(sorted_ptr),
      thrust::make_reverse_iterator(count_ptr + numel),
      thrust::make_reverse_iterator(count_ptr + numel),
      thrust::equal_to<long>(),
      thrust::maximum<long>()
    );
  }

  dim3 grid(DIVUP(numel,4), DIVUP(stride,128));
  dim3 block(32, 4);
  cunn_LookupTable_accGradParametersKernel<<<grid, block, 0, stream>>>(
    sorted_data,
    indices_data,
    THCudaTensor_data(state, gradOutput),
    THCudaTensor_data(state, gradWeight),
    count_data,
    scale,
    numel,
    stride,
    paddingValue
  );
  THCudaCheck(hipGetLastError());
}

/*
 * Keep the norm of weight smaller than maxNorm
 */
template <typename T>
struct pow_v
{
  T normType;
  pow_v(T v) : normType(v) {}
  __host__ __device__
  T operator()(const T& x) const {
    if (normType == 1)
      return std::abs(x);
    else if (normType == 2)
      return x * x;
    else
      return std::pow(std::abs(x), normType);
  }
};

template <typename T>
struct multiply_s
{
  T scale;
  multiply_s(T s) : scale(s) {}
  __host__ __device__
  T operator()(const T& x) const {
    return x * scale;
  }
};

void THNN_CudaLookupTable_renorm(
  THCState *state,
  THIndexTensor *idx,
  THCudaTensor *weight,
  float maxNorm,
  float normType)
{
  THCUNN_assertSameGPU(state, 2, idx, weight);
  if (!(THIndexTensor_(isContiguous)(state, idx) &&
        THCudaTensor_isContiguous(state, weight)))
  {
    THError("Tensors must be contiguous");
  }
  if (THIndexTensor_(nDimension)(state, idx) != 1)
    THError("idx must be a vector");
  if (normType <= 0)
    THError("non-positive-norm not supported");

  ptrdiff_t numel = THIndexTensor_(nElement)(state, idx);
  long stride = weight->stride[0];

  // get the unique indices
  thrust::device_ptr<float> weight_ptr(THCudaTensor_data(state, weight));
  thrust::device_ptr<long> idx_ptr(THIndexTensor_(data)(state, idx));
  thrust::device_ptr<long> end_ptr = thrust::unique(idx_ptr, idx_ptr+numel);
  numel = end_ptr - idx_ptr;

  pow_v<float> unary_pow(normType);
  thrust::plus<float> binary_plus;
  // numel << stride, since idx usually contains sparse row indices
  for (ptrdiff_t i = 0; i < numel; i++)
  {
    long k = idx_ptr[i] - TH_INDEX_BASE;
    thrust::device_ptr<float> row_ptr = weight_ptr + k * stride;
    float norm = thrust::transform_reduce(row_ptr, row_ptr + stride,
      unary_pow, 0, binary_plus);
    norm = std::pow(norm, (float) (1.0 / normType));
    if (norm > maxNorm)
    {
      multiply_s<float> unary_mul(maxNorm / (norm + 1e-7));
      thrust::transform(row_ptr, row_ptr + stride, row_ptr, unary_mul);
    }
  }
}
