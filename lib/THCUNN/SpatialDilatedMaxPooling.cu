#include "hip/hip_runtime.h"
#include "THCUNN.h"
#include "common.h"

// kernels borrowed from Caffe
template <typename Dtype>
__global__ void MaxPoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w, Dtype* top_data,
    Dtype* top_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + (kernel_h - 1) * dilation_h + 1, height);
    int wend = min(wstart + (kernel_w - 1) * dilation_w + 1, width);
    while(hstart < 0)
      hstart += dilation_h;
    while(wstart < 0)
      wstart += dilation_w;
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; h += dilation_h) {
      for (int w = wstart; w < wend; w += dilation_w) {
        if (bottom_data[h * width + w] > maxval) {
          maxidx = h * width + w;
          maxval = bottom_data[maxidx];
        }
      }
    }
    top_data[index] = maxval;
    top_mask[index] = maxidx + TH_INDEX_BASE;
  }
}


template <typename Dtype>
__global__ void MaxPoolBackward(const int nthreads, const Dtype* top_diff,
    const Dtype* top_mask, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart =
        (h + pad_h < ((kernel_h - 1) * dilation_h + 1)) ? 0 : (h + pad_h - ((kernel_h - 1) * dilation_h + 1)) / stride_h + 1;
    int phend = min((h + pad_h) / stride_h + 1, pooled_height);
    int pwstart =
        (w + pad_w < ((kernel_w - 1) * dilation_w + 1)) ? 0 : (w + pad_w - ((kernel_w - 1) * dilation_w + 1)) / stride_w + 1;
    int pwend = min((w + pad_w) / stride_w + 1, pooled_width);
    
    Dtype gradient = 0;
    int offset = (n * channels + c) * pooled_height * pooled_width;
    top_diff += offset;
    top_mask += offset;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
	if (top_mask[ph * pooled_width + pw] - TH_INDEX_BASE == h * width + w) {
	  gradient += top_diff[ph * pooled_width + pw];
	}
      }
    }
    bottom_diff[index] = gradient;
  }
}

void THNN_CudaSpatialDilatedMaxPooling_updateOutput(THCState *state, THCudaTensor *input, THCudaTensor *output, THCudaTensor *indices, int kW, int kH, int dW, int dH, int padW, int padH, int dilationW, int dilationH, bool ceil_mode)
{

  THCUNN_assertSameGPU(state, 3, input, output, indices);
  THArgCheck(input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  long nInputCols, nInputRows, nInputPlane, batchSize;
  long nOutputCols, nOutputRows;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  THArgCheck(nInputCols >= kW - padW && nInputRows >= kH - padH, 2, "input image smaller than kernel size");
  THArgCheck(kW/2 >= padW && kH/2 >= padH, 2, "pad should be smaller than half of kernel size");

  if(ceil_mode) {
    nOutputCols = ceil(float(nInputCols - (dilationW * (kW - 1) + 1) + 2*padW) / float(dW)) + 1;
    nOutputRows = ceil(float(nInputRows - (dilationH * (kH - 1) + 1) + 2*padH) / float(dH)) + 1;
  }
  else {
    nOutputCols = floor(float(nInputCols - (dilationW * (kW - 1) + 1) + 2*padW) / float(dW)) + 1;
    nOutputRows = floor(float(nInputRows - (dilationH * (kH - 1) + 1) + 2*padH) / float(dH)) + 1;
  }

if (nOutputCols < 1 || nOutputRows < 1)
    THError("Given input size: (%dx%dx%d). Calculated output size: (%dx%dx%d). Output size is too small",
            nInputPlane,nInputRows,nInputCols,nInputPlane,nOutputRows,nOutputCols);

if (padW || padH)
  {
    // ensure that the last pooling starts inside the image
    if ((nOutputRows - 1)*dH >= nInputRows + padH)
      --nOutputRows;
    if ((nOutputCols  - 1)*dW >= nInputCols  + padW)
      --nOutputCols;
  }

  input = THCudaTensor_newContiguous(state, input);
  float* input_data = THCudaTensor_data(state, input);

  THCudaTensor_resize4d(state, output, batchSize, nInputPlane, nOutputRows, nOutputCols);
  THCudaTensor_resizeAs(state, indices, output);

  float* indices_data = THCudaTensor_data(state, indices);
  float* output_data = THCudaTensor_data(state, output);

  int count = THCudaTensor_nElement(state, output);

  MaxPoolForward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count, input_data,
      batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
      kH, kW, dH, dW, padH, padW, dilationH, dilationW, output_data, indices_data);
  THCudaCheck(hipGetLastError());

  if(input->nDimension == 3)
    THCudaTensor_resize3d(state, output, nInputPlane, nOutputRows, nOutputCols);

  THCudaTensor_free(state, input);
}

void THNN_CudaSpatialDilatedMaxPooling_updateGradInput(THCState *state, THCudaTensor *input, THCudaTensor *gradOutput, THCudaTensor *gradInput, THCudaTensor *indices, int kW, int kH, int dW, int dH, int padW, int padH, int dilationW, int dilationH, bool ceil_mode)
{
  THCUNN_assertSameGPU(state, 4, input, gradOutput, indices, gradInput);

  input = THCudaTensor_newContiguous(state, input);
  gradOutput = THCudaTensor_newContiguous(state, gradOutput);

  long nInputCols, nInputRows, nInputPlane, batchSize;
  long nOutputCols, nOutputRows;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  if(ceil_mode) {
    nOutputCols = ceil(float(nInputCols - (dilationW * (kW - 1) + 1) + 2*padW) / float(dW)) + 1;
    nOutputRows = ceil(float(nInputRows - (dilationH * (kH - 1) + 1) + 2*padH) / float(dH)) + 1;
  }
  else {
    nOutputCols = floor(float(nInputCols - (dilationW * (kW - 1) + 1) + 2*padW) / float(dW)) + 1;
    nOutputRows = floor(float(nInputRows - (dilationH * (kH - 1) + 1) + 2*padH) / float(dH)) + 1;
  }

  if (nOutputCols < 1 || nOutputRows < 1)
    THError("Given input size: (%dx%dx%d). Calculated output size: (%dx%dx%d). Output size is too small",
            nInputPlane,nInputRows,nInputCols,nInputPlane,nOutputRows,nOutputCols);

  gradOutput = THCudaTensor_newContiguous(state, gradOutput);
  THCudaTensor_resizeAs(state, gradInput, input);

  int count = THCudaTensor_nElement(state, input);

  MaxPoolBackward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      (count,
      THCudaTensor_data(state, gradOutput),
      THCudaTensor_data(state, indices),
      batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
      kH, kW, dH, dW, padH, padW, dilationH, dilationW,
      THCudaTensor_data(state, gradInput));
  THCudaCheck(hipGetLastError());

  THCudaTensor_free(state, gradOutput);

  // clean
  THCudaTensor_free(state, input);
  THCudaTensor_free(state, gradOutput);
}
