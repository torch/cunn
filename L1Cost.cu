#include "hip/hip_runtime.h"
#include "utils.h"

#include <thrust/reduce.h>
#include <thrust/transform.h>

struct l1cost_functor
{
  l1cost_functor() {}

  __host__ __device__ float operator()(float x, float y) const
    {
      return abs(x)+abs(y);
  }
};

static int cunn_L1Cost_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

  float sum;
  long size = THCudaTensor_nElement(state, input);
  input = THCudaTensor_newContiguous(state, input);
  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  sum = thrust::reduce(input_data, input_data+size, (float) 0, l1cost_functor());

  THCudaTensor_free(state, input);

  lua_pushnumber(L, sum);
  lua_setfield(L, 1, "output");

  lua_pushnumber(L, sum);
  return 1;
}

struct l1cost_updateGradInput_functor
{
  l1cost_updateGradInput_functor() {}

  __host__ __device__ float operator()(float x) const
    {
      if(x > 0)
        return 1;
      else if(x < 0)
        return -1;
      else
        return 0;
  }
};

static int cunn_L1Cost_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

  long size = THCudaTensor_nElement(state, input);

  input = THCudaTensor_newContiguous(state, input);
  THCudaTensor_resizeAs(state, gradInput, input);

  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> gradInput_data(THCudaTensor_data(state, gradInput));

  thrust::transform(input_data, input_data+size, gradInput_data, l1cost_updateGradInput_functor());

  THCudaTensor_free(state, input);
  return 1;
}

static const struct luaL_Reg cunn_L1Cost__ [] = {
  {"L1Cost_updateOutput", cunn_L1Cost_updateOutput},
  {"L1Cost_updateGradInput", cunn_L1Cost_updateGradInput},
  {NULL, NULL}
};

static void cunn_L1Cost_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_L1Cost__, "nn");
  lua_pop(L,1);
}
