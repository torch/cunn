#include "hip/hip_runtime.h"
#include "utils.h"

#define TEMPORAL_MAX_POOLING_THREADS 1024

__global__ void cunn_TemporalMaxPooling_updateOutputKernel(float *input, float *output, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
  // Block idx is the batch index, thread idx + block idx y * MAX_THREADS is the time index
  float *input_data = input + blockIdx.x * input_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n * dW;
  float *output_data = output + blockIdx.x * output_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;
  float *indices_data = indices + blockIdx.x * output_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;

  int feat = 0;
  int time = 0;
  int max_time = input_n * kW;

  float max_value;
  float max_index = 0.0;

  if (threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS < output_w) {
    // For all features
    for (feat = 0; feat < input_n; ++feat) {
      max_value = -FLT_MAX;
      // For all values in the kernel space
      for (time = 0; time < max_time; time += input_n) {
        if (max_value < input_data[time + feat]) {
          max_value = input_data[time + feat];
          max_index = time / input_n;
        }
      }
      output_data[feat] = max_value;
      indices_data[feat] = (float)max_index;
    }
  }
}

__global__ void cunn_TemporalMaxPooling_updateGradInputKernel(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
  // Block idx is the batch index, thread idx + block idx y * MAX_THREADS is the time index
  float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n * dW;
  float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;
  float *indices_data = indices + blockIdx.x * output_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;

  int feat = 0;

  if (threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS < output_w) {
    // For all features
    for (feat = 0; feat < input_n; ++feat) {
      gradInput_data[(int)indices_data[feat] * input_n + feat] += gradOutput_data[feat];
    }
  }
}

__global__ void cunn_TemporalMaxPooling_updateGradInputKernelAtomic(float *gradInput, float *gradOutput, float *indices, int input_w, int input_n, int output_w, int kW, int dW) {
  // Block idx is the batch index, thread idx + block idx y * MAX_THREADS is the time index
  float *gradInput_data = gradInput + blockIdx.x * input_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n * dW;
  float *gradOutput_data = gradOutput + blockIdx.x * output_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;
  float *indices_data = indices + blockIdx.x * output_w * input_n + (
      threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS) * input_n;

  int feat = 0;

  if (threadIdx.x + blockIdx.y * TEMPORAL_MAX_POOLING_THREADS < output_w) {
    // For all features
    for (feat = 0; feat < input_n; ++feat) {
      atomicAdd(&gradInput_data[(int)indices_data[feat] * input_n + feat], gradOutput_data[feat]);
    }
  }
}

static int cunn_TemporalMaxPooling_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int dW = luaT_getfieldcheckint(L, 1, "dW");

  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *indices = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "indices", "torch.CudaTensor");

  int dimT = 0; // Temporal dimension
  int dimF = 1; // Feature dimension

  int batch = 1;
  int input_w;
  int input_n;
  int output_w;
  int nthreads;

  float *input_data;
  float *output_data;
  float *indices_data;

  luaL_argcheck(L, input->nDimension == 2 || input->nDimension == 3, 2, "2D or 3D(batch mode) tensor expected");

  if (input->nDimension == 3)
  {
    dimT = 1;
    dimF = 2;
    batch = input->size[0];
  }
  luaL_argcheck(L, input->size[dimT] >= kW, 2, "input sequence smaller than kernel size");

  input = THCudaTensor_newContiguous(state, input);

  input_w = input->size[dimT];
  input_n = input->size[dimF];
  output_w = (input_w - kW) / dW + 1;

  if (input->nDimension == 2)
  {
    THCudaTensor_resize2d(state, output, output_w, input->size[dimF]);
    THCudaTensor_resize2d(state, indices, output_w, input->size[dimF]);
  }
  else
  {
    THCudaTensor_resize3d(state, output, batch, output_w, input->size[dimF]);
    THCudaTensor_resize3d(state, indices, batch, output_w, input->size[dimF]);
  }

  input_data = THCudaTensor_data(state, input);
  output_data = THCudaTensor_data(state, output);
  indices_data = THCudaTensor_data(state, indices);

  dim3 blocks(batch);
  nthreads = (output_w / 32) * 32;
  if (output_w % 32 > 0) {
    nthreads += 32;
  }

  if (nthreads > TEMPORAL_MAX_POOLING_THREADS) {
    nthreads = TEMPORAL_MAX_POOLING_THREADS;
    blocks.y = nthreads / TEMPORAL_MAX_POOLING_THREADS;
    if (nthreads % TEMPORAL_MAX_POOLING_THREADS > 0) {
      blocks.y += 1;
    }
  }

  dim3 threads(nthreads);
  cunn_TemporalMaxPooling_updateOutputKernel <<< blocks, threads >>>(
      input_data, output_data, indices_data, input_w, input_n, output_w, kW, dW);

  THCudaTensor_free(state, input);

  return 1;
}

static int cunn_TemporalMaxPooling_updateGradInput(lua_State *L) {
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int dW = luaT_getfieldcheckint(L, 1, "dW");

  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  THCudaTensor *indices = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "indices", "torch.CudaTensor");

  int dimT = 0; // Temporal dimension
  int dimF = 1; // Feature dimension

  int batch = 1;
  int input_w;
  int input_n;
  int output_w;
  int nthreads;

  float *gradInput_data;
  float *gradOutput_data;
  float *indices_data;

  luaL_argcheck(L, input->nDimension == 2 || input->nDimension == 3, 2, "2D or 3D(batch mode) tensor expected");

  THCudaTensor_resizeAs(state, gradInput, input);
  THCudaTensor_zero(state, gradInput);

  if (input->nDimension == 3)
  {
    dimT = 1;
    dimF = 2;
    batch = input->size[0];
  }
  luaL_argcheck(L, input->size[dimT] >= kW, 2, "input sequence smaller than kernel size");

  gradOutput = THCudaTensor_newContiguous(state, gradOutput);

  input_w = input->size[dimT];
  input_n = input->size[dimF];
  output_w = (input_w - kW) / dW + 1;

  gradInput_data = THCudaTensor_data(state, gradInput);
  gradOutput_data = THCudaTensor_data(state, gradOutput);
  indices_data = THCudaTensor_data(state, indices);

  dim3 blocks(batch);
  nthreads = (output_w / 32) * 32;
  if (output_w % 32 > 0) {
    nthreads += 32;
  }

  if (nthreads > TEMPORAL_MAX_POOLING_THREADS) {
    nthreads = TEMPORAL_MAX_POOLING_THREADS;
    blocks.y = nthreads / TEMPORAL_MAX_POOLING_THREADS;
    if (nthreads % TEMPORAL_MAX_POOLING_THREADS > 0) {
      blocks.y += 1;
    }
  }

  dim3 threads(nthreads);
  if (kW <= dW) {
    cunn_TemporalMaxPooling_updateGradInputKernel <<< blocks, threads >>>(
        gradInput_data, gradOutput_data, indices_data, input_w, input_n, output_w, kW, dW);
  } else {
    cunn_TemporalMaxPooling_updateGradInputKernelAtomic <<< blocks, threads >>>(
        gradInput_data, gradOutput_data, indices_data, input_w, input_n, output_w, kW, dW);
  }

  THCudaTensor_free(state, gradOutput);

  return 1;
}

static const struct luaL_Reg cunn_TemporalMaxPooling__ [] = {
  {"TemporalMaxPooling_updateOutput", cunn_TemporalMaxPooling_updateOutput},
  {"TemporalMaxPooling_updateGradInput", cunn_TemporalMaxPooling_updateGradInput},
  {NULL, NULL}
};

static void cunn_TemporalMaxPooling_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_TemporalMaxPooling__, "nn");
  lua_pop(L,1);
}
