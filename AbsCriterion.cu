#include "hip/hip_runtime.h"
#include "utils.h"

#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>

struct abs_functor
{
  abs_functor() {}

  __host__ __device__ float operator()(const float& x, const float& y) const
    {
      float z = x-y;
      return z >= 0 ? z : -z;
    }
};


static int cunn_AbsCriterion_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  int sizeAverage = luaT_getfieldcheckboolean(L, 1, "sizeAverage");

  float sum;

  long size = THCudaTensor_nElement(state, input);

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  sum = thrust::inner_product(input_data, input_data+size, target_data, (float) 0, thrust::plus<float>(), abs_functor());

  if(sizeAverage)
    sum /= size;

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);

  lua_pushnumber(L, sum);
  lua_setfield(L, 1, "output");

  lua_pushnumber(L, sum);
  return 1;
}


struct abs_updateGradInput_functor
{
  const float norm;

  abs_updateGradInput_functor(float norm_) : norm(norm_) {}

  __host__ __device__ float operator()(const float& x, const float& y) const
    {
      return (x - y) >= 0 ? norm : -norm;
    }
};

static int cunn_AbsCriterion_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  int sizeAverage = luaT_getfieldcheckboolean(L, 1, "sizeAverage");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

  long size = THCudaTensor_nElement(state, input);
  float norm = (sizeAverage ? 1./size : 1.);

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

  THCudaTensor_resizeAs(state, gradInput, input);

  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  thrust::device_ptr<float> gradInput_data(THCudaTensor_data(state, gradInput));

  thrust::transform(input_data, input_data+size, target_data, gradInput_data, abs_updateGradInput_functor(norm));

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);
  return 1;
}

static const struct luaL_Reg cunn_AbsCriterion__ [] = {
  {"AbsCriterion_updateOutput", cunn_AbsCriterion_updateOutput},
  {"AbsCriterion_updateGradInput", cunn_AbsCriterion_updateGradInput},
  {NULL, NULL}
};

static void cunn_AbsCriterion_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_AbsCriterion__, "nn");
  lua_pop(L,1);
}
