#include "hip/hip_runtime.h"
#include "utils.h"

#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>
#if CUDA_VERSION >= 7000
#include <thrust/system/cuda/execution_policy.h>
#endif

struct abs_functor
{
  abs_functor() {}

  __host__ __device__ float operator()(const float& x, const float& y) const
    {
      float z = x-y;
      return z >= 0 ? z : -z;
    }
};


static int cunn_AbsCriterion_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  int sizeAverage = luaT_getfieldcheckboolean(L, 1, "sizeAverage");
  THAssert(THCudaTensor_checkGPU(state, 2, input, target));
  float sum;

  long size = THCudaTensor_nElement(state, input);

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  sum = thrust::inner_product(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    input_data, input_data+size, target_data, (float) 0,
    thrust::plus<float>(), abs_functor());

  if(sizeAverage)
    sum /= size;

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);

  lua_pushnumber(L, sum);
  lua_setfield(L, 1, "output");

  lua_pushnumber(L, sum);
  return 1;
}


struct abs_updateGradInput_functor
{
  const float norm;

  abs_updateGradInput_functor(float norm_) : norm(norm_) {}

  __host__ __device__ float operator()(const float& x, const float& y) const
    {
      return (x - y) >= 0 ? norm : -norm;
    }
};

static int cunn_AbsCriterion_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *target = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  int sizeAverage = luaT_getfieldcheckboolean(L, 1, "sizeAverage");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  THAssert(THCudaTensor_checkGPU(state, 3, input, target, gradInput));

  long size = THCudaTensor_nElement(state, input);
  float norm = (sizeAverage ? 1./size : 1.);

  input = THCudaTensor_newContiguous(state, input);
  target = THCudaTensor_newContiguous(state, target);

  THCudaTensor_resizeAs(state, gradInput, input);

  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::device_ptr<float> target_data(THCudaTensor_data(state, target));
  thrust::device_ptr<float> gradInput_data(THCudaTensor_data(state, gradInput));

  thrust::transform(
#if CUDA_VERSION >= 7000
    thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
    input_data, input_data+size, target_data, gradInput_data,
    abs_updateGradInput_functor(norm));

  THCudaTensor_free(state, input);
  THCudaTensor_free(state, target);
  return 1;
}

static const struct luaL_Reg cunn_AbsCriterion__ [] = {
  {"AbsCriterion_updateOutput", cunn_AbsCriterion_updateOutput},
  {"AbsCriterion_updateGradInput", cunn_AbsCriterion_updateGradInput},
  {NULL, NULL}
};

void cunn_AbsCriterion_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_AbsCriterion__, "nn");
  lua_pop(L,1);
}
