#include "hip/hip_runtime.h"
#include "utils.h"

struct sqrtupdateOutput_functor
{
  const double bias;

  sqrtupdateOutput_functor(double bias_) : bias(bias_) {}

  __host__ __device__ float operator()(const float& input) const
  {
    return sqrt(input+bias);
  }
};

static int cunn_Sqrt_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  double bias = luaT_getfieldchecknumber(L,1,"eps");
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  long size = THCudaTensor_nElement(state, input);

  input = THCudaTensor_newContiguous(state, input);

  THCudaTensor_resizeAs(state, output, input);

  thrust::device_ptr<float> output_data(THCudaTensor_data(state, output));
  thrust::device_ptr<float> input_data(THCudaTensor_data(state, input));
  thrust::transform(input_data, input_data+size, output_data, sqrtupdateOutput_functor(bias));

  THCudaTensor_free(state, input);
  return 1;
}

struct sqrtupdateGradInput_functor
{
  const double bias;

  sqrtupdateGradInput_functor(double bias_) : bias(bias_) {}

  __host__ __device__ float operator()(const float& output, const float& gradOutput) const
  {
    return 0.5 * gradOutput / output;
  }
};

static int cunn_Sqrt_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  double bias = luaT_getfieldchecknumber(L,1,"eps");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  long size = THCudaTensor_nElement(state, output);

  gradOutput = THCudaTensor_newContiguous(state, gradOutput);
  THCudaTensor_resizeAs(state, gradInput, output);

  thrust::device_ptr<float> output_data(THCudaTensor_data(state, output));
  thrust::device_ptr<float> gradOutput_data(THCudaTensor_data(state, gradOutput));
  thrust::device_ptr<float> gradInput_data(THCudaTensor_data(state, gradInput));
  thrust::transform(output_data, output_data+size, gradOutput_data, gradInput_data, sqrtupdateGradInput_functor(bias));

  THCudaTensor_free(state, gradOutput);
  return 1;
}

static const struct luaL_Reg cunn_Sqrt__ [] = {
  {"Sqrt_updateOutput", cunn_Sqrt_updateOutput},
  {"Sqrt_updateGradInput", cunn_Sqrt_updateGradInput},
  {NULL, NULL}
};

static void cunn_Sqrt_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_Sqrt__, "nn");
  lua_pop(L,1);
}
