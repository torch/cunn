#include "hip/hip_runtime.h"
#include "utils.h"

#ifndef DIVUP
#define DIVUP(x,y) (((x) + (y) - 1) / (y))
#endif

#define MIN(a,b) (a) < (b) ? (a) : (b)

#ifndef assert
#define assert(e)  \
    if (!(e)) { \
        printf("failed assertion `%s'\n", #e); \
        THError("aborting..."); \
    };
#endif

#include "SpatialConvolutionCUDA/updateOutput.cu"
#include "SpatialConvolutionCUDA/updateGradInput.cu"
#include "SpatialConvolutionCUDA/accGradParameters.cu"

static int cunn_SpatialConvolutionCUDA_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int padding = luaT_getfieldcheckint(L, 1, "padding");

  THCudaTensor *weight = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

  luaL_argcheck(L, input->nDimension == 4, 2, "4D (batch mode) tensor is expected");

  long nOutputPlane = weight->size[3];
  long nInputPlane  = weight->size[0];
  long kH           = weight->size[1];
  long kW           = weight->size[2];
  long inputHeight  = input->size[1];
  long inputWidth   = input->size[2];
  long batchSize    = input->size[3];
  long outputHeight = (padding + inputHeight - kH) / dH + 1;
  long outputWidth  = (padding + inputWidth - kW) / dW + 1;

  // resize output
  THCudaTensor_resize4d(state, output, nOutputPlane, outputHeight, outputWidth, batchSize);

  // asserts
  luaL_argcheck(L, inputWidth == inputHeight, 1, "input must be square");
  luaL_argcheck(L, kH == kW, 1, "kH must be equal to kW");
  luaL_argcheck(L, dH == dW, 1, "dH must be equal to dW");

  // all the data must be contiguous:
  luaL_argcheck(L, THCudaTensor_isContiguous(state, input), 2, "input must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(state, weight), 1, "weight must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(state, output), 1, "output must be contiguous");

  // raw pointers
  float *input_data = THCudaTensor_data(state, input);
  float *weight_data = THCudaTensor_data(state, weight);
  float *output_data = THCudaTensor_data(state, output);

  // convolutions
  spatialConv_updateOutput(
    input_data, weight_data, output_data,
    nInputPlane, inputHeight, inputWidth, batchSize,
    nOutputPlane, outputHeight, outputWidth,
    kH, kW,
    -floor((double)padding/2), dW,
    0, 1, true
  );

  return 1;
}

static int cunn_SpatialConvolutionCUDA_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int padding = luaT_getfieldcheckint(L, 1, "padding");

  THCudaTensor *weight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

  long nOutputPlane = weight->size[3];
  long nInputPlane  = weight->size[0];
  long kH           = weight->size[1];
  long kW           = weight->size[2];
  long inputHeight  = input->size[1];
  long inputWidth   = input->size[2];
  long batchSize    = input->size[3];
  long outputHeight = (padding + inputHeight - kH) / dH + 1;
  long outputWidth  = (padding + inputWidth - kW) / dW + 1;

  // resize gradInput
  THCudaTensor_resize4d(state, gradInput, nInputPlane, inputHeight, inputWidth, batchSize);

  // asserts
  luaL_argcheck(L, inputWidth == inputHeight, 1, "input must be square");
  luaL_argcheck(L, kH == kW, 1, "kH must be equal to kW");
  luaL_argcheck(L, dH == dW, 1, "dH must be equal to dW");

  // all the data must be contiguous:
  luaL_argcheck(L, THCudaTensor_isContiguous(state, gradInput), 2, "input must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(state, weight), 1, "weight must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(state, gradOutput), 1, "output must be contiguous");

  // raw pointers
  float *gradInput_data = THCudaTensor_data(state, gradInput);
  float *weight_data = THCudaTensor_data(state, weight);
  float *gradOutput_data = THCudaTensor_data(state, gradOutput);

  // convolutions
  spatialConv_updateGradInput(
    gradOutput_data, weight_data, gradInput_data,
    nInputPlane, inputHeight, inputWidth, batchSize,
    nOutputPlane, outputHeight, outputWidth,
    kH, kW,
    -floor((double)padding/2), dW,
    0, 1, true
  );

  return 1;
}

static int cunn_SpatialConvolutionCUDA_accGradParameters(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradWeight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");

  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int padding = luaT_getfieldcheckint(L, 1, "padding");
  int partialSum = luaT_getfieldcheckint(L, 1, "partialSum");
  float scale = luaL_optnumber(L, 4, 1);

  long nOutputPlane = gradWeight->size[3];
  long nInputPlane  = gradWeight->size[0];
  long kH           = gradWeight->size[1];
  long kW           = gradWeight->size[2];
  long inputHeight  = input->size[1];
  long inputWidth   = input->size[2];
  long batchSize    = input->size[3];
  long outputHeight = (padding + inputHeight - kH) / dH + 1;
  long outputWidth  = (padding + inputWidth - kW) / dW + 1;

  // asserts
  luaL_argcheck(L, inputWidth == inputHeight, 1, "input must be square");
  luaL_argcheck(L, kH == kW, 1, "kH must be equal to kW");
  luaL_argcheck(L, dH == dW, 1, "dH must be equal to dW");

  if (partialSum) {
    // compute partial gradients for outputHeight*outputWidth/partialSum groups of filters separately
    gradWeight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradWeightPartial", "torch.CudaTensor");
    THCudaTensor_resize4d(state, gradWeight, outputHeight*outputWidth/partialSum, nInputPlane, kH*kW, nOutputPlane);
    // numModuleY*numModulesX/partialSum, numFilterColors, filterPixels, numFilters
  }

  // all the data must be contiguous:
  luaL_argcheck(L, THCudaTensor_isContiguous(state, input), 2, "input must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(state, gradWeight), 1, "weight must be contiguous");
  luaL_argcheck(L, THCudaTensor_isContiguous(state, gradOutput), 1, "output must be contiguous");

  // raw pointers
  float *input_data = THCudaTensor_data(state, input);
  float *gradWeight_data = THCudaTensor_data(state, gradWeight);
  float *gradOutput_data = THCudaTensor_data(state, gradOutput);

  // convolutions
  spatialConv_accGradParameters(
    input_data, gradOutput_data, gradWeight_data,
    nInputPlane, inputHeight, inputWidth, batchSize,
    nOutputPlane, outputHeight, outputWidth,
    kH, kW,
    -floor((double)padding/2), dW,
    0, scale, partialSum
  );

  return 0;
}

static const struct luaL_Reg cunn_SpatialConvolutionCUDA__ [] = {
  {"SpatialConvolutionCUDA_updateOutput", cunn_SpatialConvolutionCUDA_updateOutput},
  {"SpatialConvolutionCUDA_updateGradInput", cunn_SpatialConvolutionCUDA_updateGradInput},
  {"SpatialConvolutionCUDA_accGradParameters", cunn_SpatialConvolutionCUDA_accGradParameters},
  {NULL, NULL}
};

static void cunn_SpatialConvolutionCUDA_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_SpatialConvolutionCUDA__, "nn");
  lua_pop(L,1);
}
