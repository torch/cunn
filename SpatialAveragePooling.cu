#include "hip/hip_runtime.h"
#include "utils.h"

#define CUDA_MAX_THREADS 1024   // this is safe, in reality 256 is our limit

/*
 * Description:
 *    this function avg-pools an input 3D tensor along dimensions 1 and 2
 *    3D input, 3D output
 */
__global__ void subsample(float *input, float *output,
                          int input_n, int input_h, int input_w,
                          int kH, int kW, int dH, int dW)
{
  // iterators
  int xx, yy;

  // output size
  int output_w = (input_w - kW) / dW + 1;
  int output_h = (input_h - kH) / dH + 1;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  output = output + o*output_w*output_h;
  input = input + i*input_w*input_h;

  // For all output pixels...
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {
    for(xx = xx_start; xx < xx_end; xx+=xx_step) {
      // Compute the mean of the input image...
      float *ptr_input = input + yy*dH*input_w + xx*dW;
      float *ptr_output = output + yy*output_w + xx;
      float sum = 0;
      int kx, ky;
      for(ky = 0; ky < kH; ky++) {
        for(kx = 0; kx < kW; kx++)
          sum += ptr_input[kx];
        ptr_input += input_w; // next input line
      }
      // Update output
      *ptr_output = sum;
    }
  }
}

static int cunn_SpatialAveragePooling_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");

  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

  float *output_data;
  float *input_data;

  luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  if (input->nDimension == 3) {
    long nInputCols = input->size[2];
    long nInputRows = input->size[1];
    long nOutputCols = (nInputCols - kW) / dW + 1;
    long nOutputRows = (nInputRows - kH) / dH + 1;
    long nInputPlane = input->size[0];

    luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

    input = THCudaTensor_newContiguous(state, input);
    input_data = THCudaTensor_data(state, input);

    THCudaTensor_resize3d(state, output, nInputPlane, nOutputRows, nOutputCols);
    output_data = THCudaTensor_data(state, output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    // run subsample kernel
    subsample <<<blocks, threads>>> (input_data, output_data,
                                     nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
  } else {
    long nInputCols = input->size[3];
    long nInputRows = input->size[2];
    long nbatch = input->size[0];
    long nOutputCols = (nInputCols - kW) / dW + 1;
    long nOutputRows = (nInputRows - kH) / dH + 1;
    long nInputPlane = input->size[1];

    luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

    input = THCudaTensor_newContiguous(state, input);
    input_data = THCudaTensor_data(state, input);

    THCudaTensor_resize4d(state, output, nbatch, nInputPlane, nOutputRows, nOutputCols);
    output_data = THCudaTensor_data(state, output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    // run subsample kernel
    subsample <<<blocks, threads>>> (input_data, output_data,
                                     nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
  }

  // clean
  THCudaTensor_free(state, input);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialSubsampling.updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}


/*
 * Description:
 *    this function computes the gradInput from gradOutput
 */
__global__ void subgradinput(float *gradInput, float *gradOutput,
                             int input_n, int input_h, int input_w,
                             int kH, int kW, int dH, int dW)
{
  // iterators
  int xx, yy;

  // output size
  int output_w = (input_w - kW) / dW + 1;
  int output_h = (input_h - kH) / dH + 1;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o*output_w*output_h;
  gradInput = gradInput + i*input_w*input_h;

  // compute gradInput
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {
    for(xx = xx_start; xx < xx_end; xx+=xx_step) {
      float *ptr_gradInput = gradInput + yy*dH*input_w + xx*dW;
      float *ptr_gradOutput = gradOutput + yy*output_w + xx;
      float z = *ptr_gradOutput;
      int kx, ky;
      for(ky = 0; ky < kH; ky++) {
        for(kx = 0; kx < kW; kx++)
          ptr_gradInput[kx] += z;
        ptr_gradInput += input_w;
      }
    }
  }
}


static int cunn_SpatialAveragePooling_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");

  luaL_argcheck(L, dW == kW, 1, "dW and kW must be equal (this will be fixed soon)");
  luaL_argcheck(L, dH == kH, 1, "dH and kH must be equal (this will be fixed soon)");

  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

  if (input->nDimension == 3) {
    long nInputCols = input->size[2];
    long nInputRows = input->size[1];
    long nInputPlane = input->size[0];

    float *gradOutput_data = THCudaTensor_data(state, gradOutput);
    float *gradInput_data;

    THCudaTensor_resizeAs(state, gradInput, input);
    THCudaTensor_zero(state, gradInput);
    gradInput_data = THCudaTensor_data(state, gradInput);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    // run updateGradInput kernel
    subgradinput <<<blocks, threads>>> (gradInput_data, gradOutput_data,
                                        nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
  } else {
    long nInputCols = input->size[3];
    long nInputRows = input->size[2];
    long nInputPlane = input->size[1];
    long nbatch = input->size[0];

    float *gradOutput_data = THCudaTensor_data(state, gradOutput);
    float *gradInput_data;

    THCudaTensor_resizeAs(state, gradInput, input);
    THCudaTensor_zero(state, gradInput);
    gradInput_data = THCudaTensor_data(state, gradInput);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    // run updateGradInput kernel
    subgradinput <<<blocks, threads>>> (gradInput_data, gradOutput_data,
                                        nInputPlane, nInputRows, nInputCols, kH, kW, dH, dW);
  }

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialSubsampling.updateGradInput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static const struct luaL_Reg cunn_SpatialAveragePooling__ [] = {
  {"SpatialAveragePooling_updateOutput", cunn_SpatialAveragePooling_updateOutput},
  {"SpatialAveragePooling_updateGradInput", cunn_SpatialAveragePooling_updateGradInput},
  {NULL, NULL}
};

static void cunn_SpatialAveragePooling_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_SpatialAveragePooling__, "nn");
  lua_pop(L,1);
}

#undef CUDA_MAX_THREADS
