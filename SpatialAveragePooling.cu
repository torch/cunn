#include "hip/hip_runtime.h"
#include "utils.h"

#define CUDA_MAX_THREADS 1024   // this is safe, in reality 256 is our limit

/*
 * Description:
 *    this function avg-pools an input 3D tensor along dimensions 1 and 2
 *    3D input, 3D output
 */
__global__ void subsample(float *input, float *output,
                          int input_n, int input_h, int input_w,
                          int output_h, int output_w,
                          int kH, int kW, int dH, int dW)
{
  // iterators
  int xx, yy;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  output = output + o*output_w*output_h;
  input = input + i*input_w*input_h;

  // For all output pixels...
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {
    for(xx = xx_start; xx < xx_end; xx+=xx_step) {
      // Get effective pooling window size
      int hend = min(kH, input_h - yy*dH);
      int wend = min(kW, input_w - xx*dW);

      // Compute the mean of the input image...
      float *ptr_input = input + yy*dH*input_w + xx*dW;
      float *ptr_output = output + yy*output_w + xx;
      float sum = 0;
      int kx, ky;
      for(ky = 0; ky < hend; ky++) {
        for(kx = 0; kx < wend; kx++)
          sum += ptr_input[kx];
        ptr_input += input_w; // next input line
      }
      // Update output
      *ptr_output = sum/float(wend*hend);
    }
  }
}

static int cunn_SpatialAveragePooling_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  bool ceil_mode = luaT_getfieldcheckboolean(L, 1, "ceil_mode");

  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THAssert(THCudaTensor_checkGPU(state, 2, input, output));

  float *output_data;
  float *input_data;

  luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch) tensor expected");

  long nOutputCols;
  long nOutputRows;

  if (input->nDimension == 3) {
    long nInputCols = input->size[2];
    long nInputRows = input->size[1];
    long nInputPlane = input->size[0];

    if(ceil_mode) {
      nOutputCols = ceil(float(nInputCols - kW) / float(dW)) + 1;
      nOutputRows = ceil(float(nInputRows - kH) / float(dH)) + 1;
    }
    else {
      nOutputCols = floor(float(nInputCols - kW) / float(dW)) + 1;
      nOutputRows = floor(float(nInputRows - kH) / float(dH)) + 1;
    }

    luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

    input = THCudaTensor_newContiguous(state, input);
    input_data = THCudaTensor_data(state, input);

    THCudaTensor_resize3d(state, output, nInputPlane, nOutputRows, nOutputCols);
    output_data = THCudaTensor_data(state, output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    // run subsample kernel
    subsample <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (input_data, output_data,
                                     nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
                                     kH, kW, dH, dW);
  } else {
    long nInputCols = input->size[3];
    long nInputRows = input->size[2];
    long nInputPlane = input->size[1];
    long nbatch = input->size[0];

    if(ceil_mode) {
      nOutputCols = ceil(float(nInputCols - kW) / float(dW)) + 1;
      nOutputRows = ceil(float(nInputRows - kH) / float(dH)) + 1;
    }
    else {
      nOutputCols = floor(float(nInputCols - kW) / float(dW)) + 1;
      nOutputRows = floor(float(nInputRows - kH) / float(dH)) + 1;
    }

    luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

    input = THCudaTensor_newContiguous(state, input);
    input_data = THCudaTensor_data(state, input);

    THCudaTensor_resize4d(state, output, nbatch, nInputPlane, nOutputRows, nOutputCols);
    output_data = THCudaTensor_data(state, output);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    // run subsample kernel
    subsample <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (input_data, output_data,
                                     nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
                                     kH, kW, dH, dW);
  }

  // clean
  THCudaTensor_free(state, input);

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialAveragePooling.updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}


/*
 * Description:
 *    this function computes the gradInput from gradOutput
 */
__global__ void subgradinput(float *gradInput, float *gradOutput,
                             int input_n, int input_h, int input_w,
                             int output_h, int output_w,
                             int kH, int kW, int dH, int dW)
{
  // iterators
  int xx, yy;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o*output_w*output_h;
  gradInput = gradInput + i*input_w*input_h;

  // compute gradInput
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {
    for(xx = xx_start; xx < xx_end; xx+=xx_step) {
      // Get effective pooling window size
      int hend = min(kH, input_h - yy*dH);
      int wend = min(kW, input_w - xx*dW);

      float *ptr_gradInput = gradInput + yy*dH*input_w + xx*dW;
      float *ptr_gradOutput = gradOutput + yy*output_w + xx;
      float z = *ptr_gradOutput;
      int kx, ky;
      for(ky = 0; ky < hend; ky++) {
        for(kx = 0; kx < wend; kx++)
          ptr_gradInput[kx] += z / float(wend*hend);
        ptr_gradInput += input_w;
      }
    }
  }
}

/*
 * Description:
 *    this function computes the gradInput from gradOutput
 *    but with an atomic accumulation. It is needed to be done so
 *    for cases of kH != dH and kW != dW
 */
__global__ void subgradinputAtomic(float *gradInput, float *gradOutput,
                                   int input_n, int input_h, int input_w,
                                   int output_h, int output_w,
                                   int kH, int kW, int dH, int dW)
{
  // iterators
  int xx, yy;

  // compute offsets based on thread/block ID
  int o = blockIdx.x;
  int i = o;

  int xx_start = threadIdx.x;
  int xx_end = output_w;
  int xx_step = blockDim.x;

  int yy_start = blockDim.y*blockIdx.y + threadIdx.y;
  int yy_end = output_h;
  int yy_step = blockDim.y*gridDim.y;

  // select input/output plane
  gradOutput = gradOutput + o*output_w*output_h;
  gradInput = gradInput + i*input_w*input_h;

  // compute gradInput
  for(yy = yy_start; yy < yy_end; yy+=yy_step) {
    for(xx = xx_start; xx < xx_end; xx+=xx_step) {
      // Get effective pooling window size
      int hend = min(kH, input_h - yy*dH);
      int wend = min(kW, input_w - xx*dW);

      float *ptr_gradInput = gradInput + yy*dH*input_w + xx*dW;
      float *ptr_gradOutput = gradOutput + yy*output_w + xx;
      float z = *ptr_gradOutput;
      int kx, ky;
      for(ky = 0; ky < hend; ky++) {
        for(kx = 0; kx < wend; kx++) {
          atomicAdd(&(ptr_gradInput[kx]), z / float(wend*hend));
        }
        ptr_gradInput += input_w;
      }
    }
  }
}


static int cunn_SpatialAveragePooling_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  bool ceil_mode = luaT_getfieldcheckboolean(L, 1, "ceil_mode");

  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  THAssert(THCudaTensor_checkGPU(state, 3, input, gradInput, gradOutput));

  long nOutputCols;
  long nOutputRows;

  if (input->nDimension == 3) {
    long nInputCols = input->size[2];
    long nInputRows = input->size[1];
    long nInputPlane = input->size[0];

    if(ceil_mode) {
      nOutputCols = ceil(float(nInputCols - kW) / float(dW)) + 1;
      nOutputRows = ceil(float(nInputRows - kH) / float(dH)) + 1;
    }
    else {
      nOutputCols = floor(float(nInputCols - kW) / float(dW)) + 1;
      nOutputRows = floor(float(nInputRows - kH) / float(dH)) + 1;
    }

    float *gradOutput_data = THCudaTensor_data(state, gradOutput);
    float *gradInput_data;

    THCudaTensor_resizeAs(state, gradInput, input);
    THCudaTensor_zero(state, gradInput);
    gradInput_data = THCudaTensor_data(state, gradInput);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane,yblocks);
    dim3 threads(32,8);

    // run updateGradInput kernel
    if (kH == dH && kW == dW) {
      subgradinput <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data,
                                          nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
                                          kH, kW, dH, dW);
    } else {
      subgradinputAtomic <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data,
                                                nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
                                                kH, kW, dH, dW);
    }
  } else {
    long nInputCols = input->size[3];
    long nInputRows = input->size[2];
    long nInputPlane = input->size[1];
    long nbatch = input->size[0];

    if(ceil_mode) {
      nOutputCols = ceil(float(nInputCols - kW) / float(dW)) + 1;
      nOutputRows = ceil(float(nInputRows - kH) / float(dH)) + 1;
    }
    else {
      nOutputCols = floor(float(nInputCols - kW) / float(dW)) + 1;
      nOutputRows = floor(float(nInputRows - kH) / float(dH)) + 1;
    }

    float *gradOutput_data = THCudaTensor_data(state, gradOutput);
    float *gradInput_data;

    THCudaTensor_resizeAs(state, gradInput, input);
    THCudaTensor_zero(state, gradInput);
    gradInput_data = THCudaTensor_data(state, gradInput);

    // cuda blocks & threads:
    int yblocks = (int)(16L / nInputPlane);
    yblocks = yblocks < 1 ? 1 : yblocks;
    dim3 blocks(nInputPlane*nbatch,yblocks);
    dim3 threads(32,8);

    // run updateGradInput kernel
    if (kH == dH && kW == dW) {
      subgradinput <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data,
                                          nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
                                          kH, kW, dH, dW);
    } else {
      subgradinputAtomic <<<blocks, threads, 0, THCState_getCurrentStream(state)>>> (gradInput_data, gradOutput_data,
                                                nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
                                                kH, kW, dH, dW);
    }
  }

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialAveragePooling.updateGradInput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static const struct luaL_Reg cunn_SpatialAveragePooling__ [] = {
  {"SpatialAveragePooling_updateOutput", cunn_SpatialAveragePooling_updateOutput},
  {"SpatialAveragePooling_updateGradInput", cunn_SpatialAveragePooling_updateGradInput},
  {NULL, NULL}
};

void cunn_SpatialAveragePooling_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_SpatialAveragePooling__, "nn");
  lua_pop(L,1);
}

#undef CUDA_MAX_THREADS
