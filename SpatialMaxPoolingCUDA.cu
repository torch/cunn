#include "utils.h"

#include "SpatialPoolingCUDA/updateOutput.cu"
#include "SpatialPoolingCUDA/updateGradInput.cu"

static int cunn_SpatialMaxPoolingCUDA_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");

  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

  luaL_argcheck(L, input->nDimension == 4, 2, "4D (batch) tensor expected");

  long nInputCols = input->size[2];
  long nInputRows = input->size[1];
  long nInputPlane = input->size[0];
  long batchSize = input->size[3];
  long nOutputCols = (nInputCols - kW) / dW + 1;
  long nOutputRows = (nInputRows - kH) / dH + 1;

  luaL_argcheck(L, nInputCols >= kW && nInputRows >= kH, 2, "input image smaller than kernel size");

  luaL_argcheck(L, THCudaTensor_isContiguous(state, input), 2, "input must be contiguous");
  float *input_data = THCudaTensor_data(state, input);

  THCudaTensor_resize4d(state, output, nInputPlane, nOutputRows, nOutputCols, batchSize);
  float *output_data = THCudaTensor_data(state, output);

  spatialMaxPooling_updateOutput<MaxPooler>
    (input_data, output_data,
     nInputPlane, nInputRows, nInputCols, batchSize,
     nOutputRows, nOutputCols,
     kH, kW,
     0, dW);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialMaxPoolingCUDA.updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static int cunn_SpatialMaxPoolingCUDA_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");

  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

  long nInputCols = input->size[2];
  long nInputRows = input->size[1];
  long nInputPlane = input->size[0];
  long batchSize = input->size[3];
  long nOutputCols = (nInputCols - kW) / dW + 1;
  long nOutputRows = (nInputRows - kH) / dH + 1;

  THCudaTensor_resizeAs(state, gradInput, input);
  THCudaTensor_zero(state, gradInput);

  float *input_data = THCudaTensor_data(state, input);
  float *output_data = THCudaTensor_data(state, output);
  float *gradOutput_data = THCudaTensor_data(state, gradOutput);
  float *gradInput_data = THCudaTensor_data(state, gradInput);

  spatialMaxPooling_updateGradInput
    (input_data, gradOutput_data, output_data, gradInput_data,
     nInputPlane, nInputRows, nInputCols, batchSize,
     nOutputRows, nOutputCols,
     kH, kW,
     0, dW);

  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in SpatialMaxPoolingCUDA.updateGradInput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
  return 1;
}

static const struct luaL_Reg cunn_SpatialMaxPoolingCUDA__ [] = {
  {"SpatialMaxPoolingCUDA_updateOutput", cunn_SpatialMaxPoolingCUDA_updateOutput},
  {"SpatialMaxPoolingCUDA_updateGradInput", cunn_SpatialMaxPoolingCUDA_updateGradInput},
  {NULL, NULL}
};

static void cunn_SpatialMaxPoolingCUDA_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_SpatialMaxPoolingCUDA__, "nn");
  lua_pop(L,1);
}
