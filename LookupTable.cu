#include "hip/hip_runtime.h"
#include "utils.h"
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/constant_iterator.h>
#if CUDA_VERSION >= 7000
#include <thrust/system/cuda/execution_policy.h>
#endif

#ifndef DIVUP
#define DIVUP(x, y) (((x) + (y) - 1) / (y))
#endif

const int WARP_SIZE = 32;

__device__ __forceinline__ bool warpHasCollision(int val) {
  // Compare our value to the values stored in the next 16 lanes,
  // wrapping around at 32. If any pair of values is the same than
  // there is a collision in the warp.
  bool dup = 0;
  const int laneId = threadIdx.x % 32;

#if __CUDA_ARCH__ >= 300
  #pragma unroll
  for (int i = 1; i <= 16; i++) {
    dup |= (__shfl(val, (laneId + i) % 32) == val);
  }
#else
  volatile __shared__ int values[128];
  values[threadIdx.x] = val;
  const int offset = threadIdx.x - laneId;

  #pragma unroll
  for (int i = 1; i <= 16; i++) {
    dup |= (values[offset + ((laneId + i) % 32)] == val);
  }
#endif

  return __any(dup) != 0;
}

__global__ void cunn_LookupTable_accGradParametersKernelByFeature(
  float *input, float *gradOutput, float *gradWeight, float scale, long numel, long stride) {

  const int featureDim = blockIdx.x * 4 + threadIdx.x / 32;
  if (featureDim >= stride) {
    return;
  }

  // The strategy here is that each warp handles a single feature
  // dimension.
  // Within that feature dimension, points in the [batch][element]
  // dimension can overlap, and we need to determine if threads want
  // to add to the gradient in a colliding manner.
  // Typically one would use floating-point atomicAdd() to resolve
  // these collisions, but that is non-deterministic if there are
  // collisions. Non-determinism for this code is really bad,
  // especially in RNNs, and is prone to snowballing error.
  // In order to get a deterministic order of execution, we handle
  // non-colliding updates separately from colliding ones. Colliding
  // updates are serialized in their order of execution by using the
  // warp-wide collision detector `warpHasCollision`.
  const int laneId = threadIdx.x % 32;
  for (int i = laneId; i < numel; i += WARP_SIZE) {
    int weightIndex = (int) (input[i] - 1);
    float update = gradOutput[i*stride + featureDim] * scale;

    // Check for collision
    if (warpHasCollision(weightIndex)) {
      // Run all lanes sequentially; warp divergence
      for (int i = 0; i < WARP_SIZE; ++i) {
        if (laneId == i) {
          gradWeight[weightIndex*stride + featureDim] += update;
        }
      }
    } else {
      // No collision; warp coherence
      gradWeight[weightIndex*stride + featureDim] += update;
    }
  }
}

__global__ void cunn_LookupTable_accGradParametersKernel(
  float *input, float *indices, float *gradOutput, float *gradWeight, float *count, float defaultScale, long numel, long stride) {

  int idx = blockIdx.x * 4 + threadIdx.y;

  // Each warp is responsible for an input into the LookupTable.
  // If the preceeding input has the same as this input, then the warp
  // exits immediately. The warp also processes subsequent inputs with the
  // same value.
  //
  // Input Warp
  // 1     <warp 1>
  // 1     <warp 1> (<warp 2> exits without doing any work)
  // 5     <warp 3>
  // 8     <warp 4>

  // Number of values proceessed by each thread (grain size)
  const int SZ = 4;

  if (idx < numel && (idx == 0 || input[idx] != input[idx - 1])) {
    do {
      const int startFeature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
      const int weightRow = ((int) input[idx] - 1) * stride;
      const int gradOutputRow = ((int) indices[idx] - 1) * stride;
      const float scale = count ? defaultScale / count[idx] : defaultScale;

      float gradient[SZ];
      float weight[SZ];

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++) {
        int featureDim = startFeature + ii * WARP_SIZE;
        if (featureDim < stride) {
          gradient[ii] = gradOutput[gradOutputRow + featureDim];
          weight[ii] = gradWeight[weightRow + featureDim];
        }
      }

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++) {
        weight[ii] += gradient[ii] * scale;
      }

      #pragma unroll
      for (int ii = 0; ii < SZ; ii++) {
        int featureDim = startFeature + ii * WARP_SIZE;
        if (featureDim < stride) {
          gradWeight[weightRow + featureDim] = weight[ii];
        }
      }

      idx++;
    } while (idx < numel && input[idx] == input[idx - 1]);
  }
}

static int cunn_LookupTable_accGradParameters(lua_State *L)
{
  THCState* state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*) luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor*) luaT_checkudata(L, 3, "torch.CudaTensor");
  float scale = luaL_optnumber(L, 4, 1);
  THCudaTensor *gradWeight = (THCudaTensor*) luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
  THCudaTensor *sorted = (THCudaTensor*) luaT_getfieldcheckudata(L, 1, "_sorted", "torch.CudaTensor");
  THCudaTensor *indices = (THCudaTensor*) luaT_getfieldcheckudata(L, 1, "_indices", "torch.CudaTensor");
  THCudaTensor *count = (THCudaTensor*) luaT_getfieldcheckudata(L, 1, "_count", "torch.CudaTensor");
  bool scaleGradByFreq = luaT_getfieldcheckboolean(L, 1, "shouldScaleGradByFreq");

  THAssert(THCudaTensor_checkGPU(state, 5, input, gradOutput, gradWeight, sorted, indices));
  if (!(THCudaTensor_isContiguous(state, input) &&
        THCudaTensor_isContiguous(state, gradOutput) &&
        THCudaTensor_isContiguous(state, gradWeight))) {
    luaL_error(L, "Tensors must be contiguous");
  }

  int nDim = THCudaTensor_nDimension(state, input);
  if (nDim != 1 && nDim != 2)
    luaL_error(L, "input must be a vector or matrix");

  long numel = THCudaTensor_nElement(state, input);
  long stride = gradWeight->stride[0];

  hipStream_t stream = THCState_getCurrentStream(state);

  if (numel <= 768 && !scaleGradByFreq) {
    cunn_LookupTable_accGradParametersKernelByFeature<<<DIVUP(stride,4), 128, 0, stream>>>(
      THCudaTensor_data(state, input),
      THCudaTensor_data(state, gradOutput),
      THCudaTensor_data(state, gradWeight),
      scale,
      numel,
      stride);

    return 0;
  }

  THCudaTensor_resizeAs(state, sorted, input);
  THCudaTensor_resizeAs(state, indices, input);

  // Sort the inputs into sorted with the corresponding indices
  THCudaTensor_sort(state, sorted, indices, input, 0, 0);

  float *sorted_data = THCudaTensor_data(state, sorted);
  float *indices_data = THCudaTensor_data(state, indices);
  float *count_data = NULL;

  if (scaleGradByFreq) {
    THCudaTensor_resizeAs(state, count, input);
    count_data = THCudaTensor_data(state, count);

    thrust::device_ptr<float> sorted_ptr(sorted_data);
    thrust::device_ptr<float> count_ptr(count_data);

    // Compute an increasing sequence per unique item in sorted:
    // sorted: 2 5 5 5 7 7 8 9 9
    //  count: 1 1 2 3 1 2 1 1 2
    thrust::inclusive_scan_by_key(
#if CUDA_VERSION >= 7000
      thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
      sorted_ptr,
      sorted_ptr + numel,
      thrust::make_constant_iterator(1),
      count_ptr);

    // Take the maximum of each count per unique key in reverse:
    // sorted: 2 5 5 5 7 7 8 9 9
    //  count: 1 3 3 3 2 2 1 2 2
    thrust::inclusive_scan_by_key(
#if CUDA_VERSION >= 7000
      thrust::cuda::par.on(THCState_getCurrentStream(state)),
#endif
      thrust::make_reverse_iterator(sorted_ptr + numel),
      thrust::make_reverse_iterator(sorted_ptr),
      thrust::make_reverse_iterator(count_ptr + numel),
      thrust::make_reverse_iterator(count_ptr + numel),
      thrust::equal_to<float>(),
      thrust::maximum<float>());
  }

  dim3 grid(DIVUP(numel,4), DIVUP(stride,128));
  dim3 block(32, 4);
  cunn_LookupTable_accGradParametersKernel<<<grid, block, 0, stream>>>(
    sorted_data,
    indices_data,
    THCudaTensor_data(state, gradOutput),
    THCudaTensor_data(state, gradWeight),
    count_data,
    scale,
    numel,
    stride);

  return 0;
}

static const struct luaL_Reg cunn_LookupTable__ [] = {
  {"LookupTable_accGradParameters", cunn_LookupTable_accGradParameters},
  {NULL, NULL}
};

void cunn_LookupTable_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_LookupTable__, "nn");
  lua_pop(L,1);
}
