#include "hip/hip_runtime.h"
#include "utils.h"

struct sigmoidupdateOutput_functor
{
  __device__ void operator()(float* output, const float* input) const
  {
    *output = 1./(1.+ exp(-*input));
  }
};

static int cunn_Sigmoid_updateOutput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

  THCudaTensor_resizeAs(state, output, input);
  THCudaTensor_pointwiseApply2(state, output, input, sigmoidupdateOutput_functor());
  return 1;
}

struct sigmoidupdateGradInput_functor
{
  __device__ void operator()(float* gradInput, const float* output, const float* gradOutput) const
  {
    *gradInput = *gradOutput * (1.-*output) * *output;
  }
};

static int cunn_Sigmoid_updateGradInput(lua_State *L)
{
  THCState *state = getCutorchState(L);
  THCudaTensor *output = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

  THCudaTensor_resizeAs(state, gradInput, output);
  THCudaTensor_pointwiseApply3(state, gradInput, output, gradOutput, sigmoidupdateGradInput_functor());
  return 1;
}

static const struct luaL_Reg cunn_Sigmoid__ [] = {
  {"Sigmoid_updateOutput", cunn_Sigmoid_updateOutput},
  {"Sigmoid_updateGradInput", cunn_Sigmoid_updateGradInput},
  {NULL, NULL}
};

static void cunn_Sigmoid_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_Sigmoid__, "nn");
  lua_pop(L,1);
}
