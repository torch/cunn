#include "hip/hip_runtime.h"
#include "utils.h"

struct MaxFloat {
  __device__ __forceinline__ float operator()(float max, float v) const {
    return fmaxf(max, v);
  }
};

struct SumFloat {
  __device__ __forceinline__ float operator()(float sum, float v) const {
    return sum + v;
  }
};

struct SumExpFloat {
  __device__ __forceinline__ SumExpFloat(float v) : max_k(v) {}

  __device__ __forceinline__ float operator()(float sum, float v) const {
    return sum + expf(v - max_k);
  }

  const float max_k;
};

struct NoFinal {
  __device__ __forceinline__ float operator()(float v) const {
    return v;
  }
};

struct LSMFinal {
  __device__ __forceinline__ LSMFinal(float m) : max_k(m) {}

  __device__ __forceinline__ float operator()(float v) const {
    return max_k + logf(v);
  }

  const float max_k;
};

template <typename Reduction, typename Finalize>
__device__ __forceinline__ float
blockReduce(float* smem, float val,
            const Reduction& r,
            float defaultVal,
            const Finalize& f) {
  // To avoid RaW races from chaning blockReduce calls together, we
  // need a sync here
  __syncthreads();

  smem[threadIdx.x] = val;

  __syncthreads();

  float warpVal = defaultVal;

  // First warp will perform per-warp reductions for the remaining warps
  if ((threadIdx.x / 32) == 0) {
    int lane = threadIdx.x % 32;

    if (lane < blockDim.x / 32) {
#pragma unroll
      for (int i = 0; i < 32; ++i) {
        warpVal = r(warpVal, smem[lane * 32 + i]);
      }

      smem[lane] = warpVal;
    }
  }

  __syncthreads();

  // First thread will perform a reduction of the above per-warp reductions
  float blockVal = defaultVal;

  if (threadIdx.x == 0) {
    for (int i = 0; i < blockDim.x / 32; ++i) {
      blockVal = r(blockVal, smem[i]);
    }

    smem[0] = f(blockVal);
  }

  // Sync and broadcast
  __syncthreads();
  return smem[0];
}

template <typename Reduction>
__device__ __forceinline__ float
blockReduce(float* smem, float val,
            const Reduction& r,
            float defaultVal) {
  return blockReduce<Reduction, NoFinal>(smem, val, r, defaultVal, NoFinal());
}

template <typename Reduction, int ILP>
__device__ __forceinline__ float
ilpReduce(float* data,
          int size,
          const Reduction& r,
          float defaultVal) {
  float threadVal = defaultVal;
  int offset = threadIdx.x;

  int last = size % (ILP * blockDim.x);

  // Body (unroll by ILP times)
  for (; offset < size - last;
       offset += blockDim.x * ILP) {
  float tmp[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      tmp[j] = data[offset + j * blockDim.x];
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      threadVal = r(threadVal, tmp[j]);
    }
  }

  // Epilogue
  for (; offset < size; offset += blockDim.x) {
    threadVal = r(threadVal, data[offset]);
  }

  return threadVal;
}

template <int ILP>
__global__ void
cunn_LogSoftMax_updateOutput_kernel(float *output, float *input, int classes) {
  extern __shared__ float buffer[];
  input += blockIdx.x * classes;
  output += blockIdx.x * classes;

  float threadMax =
    ilpReduce<MaxFloat, ILP>(input, classes, MaxFloat(), -FLT_MAX);
  float max_k =
    blockReduce<MaxFloat>(buffer, threadMax, MaxFloat(), -FLT_MAX);

  float threadExp =
    ilpReduce<SumExpFloat, ILP>(input, classes, SumExpFloat(max_k), 0.0f);
  float logsum_k =
    blockReduce<SumFloat, LSMFinal>(
      buffer, threadExp, SumFloat(), 0.0f, LSMFinal(max_k));

  // Output LSM (hand ILP)
  int offset = threadIdx.x;

  int last = classes % (ILP * blockDim.x);
  for ( ; offset < classes - last; offset += blockDim.x * ILP) {
    float tmp[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      tmp[j] = input[offset + j * blockDim.x];
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      output[offset + j * blockDim.x] = tmp[j] - logsum_k;
    }
  }

  for (; offset < classes; offset += blockDim.x) {
    output[offset] = input[offset] - logsum_k;
  }
}

template <int ILP>
__global__ void
cunn_LogSoftMax_updateGradInput_kernel(float *gradInput,
                                       float *output,
                                       float *gradOutput,
                                       int classes) {
  extern __shared__ float buffer[];
  gradInput += blockIdx.x * classes;
  output += blockIdx.x * classes;
  gradOutput += blockIdx.x * classes;

  float threadSum =
    ilpReduce<SumFloat, 4>(gradOutput, classes, SumFloat(), 0.0f);
  float sum_k =
    blockReduce<SumFloat>(buffer, threadSum, SumFloat(), 0.0f);

  // Update gradInput (hand ILP)
  int offset = threadIdx.x;
  int last = classes % (ILP * blockDim.x);
  for ( ; offset < classes - last; offset += blockDim.x * ILP) {
    float tmpGradOutput[ILP];
    float tmpOutput[ILP];

#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      tmpGradOutput[j] = gradOutput[offset + j * blockDim.x];
      tmpOutput[j] = output[offset + j * blockDim.x];
    }

#pragma unroll
    for (int j = 0; j < ILP; ++j) {
      gradInput[offset + j * blockDim.x] =
        tmpGradOutput[j] - __expf(tmpOutput[j]) * sum_k;
    }
  }

  for (; offset < classes; offset += blockDim.x) {
    gradInput[offset] =
      gradOutput[offset] - __expf(output[offset]) * sum_k;
  }
}

static int cunn_LogSoftMax_updateOutput(lua_State *L) {
  THCState *state = getCutorchState(L);
  THCudaTensor *input =
    (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *output =
    (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

  input = THCudaTensor_newContiguous(state, input);
  THCudaTensor_resizeAs(state, output, input);

  int batchSize = 1;
  int classSize = 0;

  if (THCudaTensor_nDimension(state, input) == 1) {
    classSize = THCudaTensor_size(state, input, 0);
  } else if (THCudaTensor_nDimension(state, input) == 2) {
    batchSize = THCudaTensor_size(state, input, 0);
    classSize = THCudaTensor_size(state, input, 1);
  } else {
    THError("vector or matrix expected");
  }

  dim3 grid(batchSize);
  dim3 block(1024);

  cunn_LogSoftMax_updateOutput_kernel<2>
    <<<grid, block, block.x * sizeof(float)>>>(
      THCudaTensor_data(state, output),
      THCudaTensor_data(state, input),
      classSize);

  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess) {
    THError(hipGetErrorString(errcode));
  }

  THCudaTensor_free(state, input);
  return 1;
}

static int cunn_LogSoftMax_updateGradInput(lua_State *L) {
  THCState *state = getCutorchState(L);
  THCudaTensor *gradOutput =
    (THCudaTensor*)luaT_checkudata(L, 3, "torch.CudaTensor");
  THCudaTensor *output =
    (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");
  THCudaTensor *gradInput =
    (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

  output = THCudaTensor_newContiguous(state, output);
  gradOutput = THCudaTensor_newContiguous(state, gradOutput);

  THCudaTensor_resizeAs(state, gradInput, output);

  int batchSize = 1;
  int classSize = 0;

  if (THCudaTensor_nDimension(state, gradInput) == 1) {
    classSize = THCudaTensor_size(state, gradInput, 0);
  } else if (THCudaTensor_nDimension(state, gradInput) == 2) {
    batchSize = THCudaTensor_size(state, gradInput, 0);
    classSize = THCudaTensor_size(state, gradInput, 1);
  } else {
    THError("vector or matrix expected");
  }

  dim3 grid(batchSize);
  dim3 block(1024);

  cunn_LogSoftMax_updateGradInput_kernel<2>
    <<<grid, block, block.x * sizeof(float)>>>(
      THCudaTensor_data(state, gradInput),
      THCudaTensor_data(state, output),
      THCudaTensor_data(state, gradOutput),
      classSize);

  hipError_t errcode = hipGetLastError();
  if (errcode != hipSuccess) {
    THError(hipGetErrorString(errcode));
  }

  THCudaTensor_free(state, gradOutput);
  THCudaTensor_free(state, output);
  return 1;
}

static const struct luaL_Reg cunn_LogSoftMax__ [] = {
  {"LogSoftMax_updateOutput", cunn_LogSoftMax_updateOutput},
  {"LogSoftMax_updateGradInput", cunn_LogSoftMax_updateGradInput},
  {NULL, NULL}
};

static void cunn_LogSoftMax_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_LogSoftMax__, "nn");
  lua_pop(L,1);
}
