#include "utils.h"
#include "im2col.h"


int cunn_SpatialDeconvolution_updateOutput(lua_State *L) {
  THCState *state = getCutorchState(L);
  // Input
  THCudaTensor *input = (THCudaTensor*)luaT_checkudata(L, 2, "torch.CudaTensor");

  // Params:
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int nOutputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");
  int nInputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
  int padW = luaT_getfieldcheckint(L, 1, "padW");
  int padH = luaT_getfieldcheckint(L, 1, "padH");

  THCudaTensor *weight  = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *bias    = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "bias", "torch.CudaTensor");
  THCudaTensor *columns = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "finput", "torch.CudaTensor");
  THCudaTensor *ones    = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "fgradInput", "torch.CudaTensor");
  THCudaTensor *output  = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "output", "torch.CudaTensor");

  THAssert(THCudaTensor_checkGPU(state, 6, input, output, weight,
                                 bias, columns, ones));
  luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch mode) tensor is expected");

  int batch = 1;
  if (input->nDimension == 3) {
    luaL_argcheck(L, input->size[0] == nInputPlane, 2, "input channels and nInputPlane dont match");
    // Force batch
    batch = 0;
    THCudaTensor_resize4d(state, input, 1, input->size[0], input->size[1], input->size[2]);
  } else {
    luaL_argcheck(L, input->size[1] == nInputPlane, 2, "input channels and nInputPlane dont match");
  }

  long inputWidth   = input->size[3];
  long inputHeight  = input->size[2];
  long outputWidth  = (inputWidth - 1) * dW - 2*padW + kW;
  long outputHeight = (inputHeight - 1) * dH - 2*padH + kH;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCudaTensor_resize4d(state, output, batchSize, nOutputPlane, outputHeight, outputWidth);

  // Resize temporary columns
  THCudaTensor_resize2d(state, columns, nOutputPlane*kW*kH, inputHeight*inputWidth);

  // Define a buffer of ones, for bias accumulation
  // Note: this buffer can be shared with other modules, it only ever gets increased,
  // and always contains ones.
  if (ones->nDimension != 2 || ones->size[0]*ones->size[1] < outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCudaTensor_resize2d(state, ones, outputHeight, outputWidth);
    THCudaTensor_fill(state, ones, 1);
  }

  // Helpers
  THCudaTensor *input_n = THCudaTensor_new(state);
  THCudaTensor *output_n = THCudaTensor_new(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCudaTensor_select(state, input_n, input, 0, elt);
    THCudaTensor_select(state, output_n, output, 0, elt);

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = weight->size[1];
    long n = columns->size[1];
    long k = weight->size[0];

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    THCudaBlas_gemm(
        state,
        'n', 't',
        n, m, k,
        1,
        THCudaTensor_data(state, input_n), n,
        THCudaTensor_data(state, weight), m,
        0,
        THCudaTensor_data(state, columns), n
    );

    // Unpack columns back into input:
    col2im(
      THCState_getCurrentStream(state),
      THCudaTensor_data(state, columns),
      nOutputPlane, outputHeight, outputWidth, kH, kW, padH, padW, dH, dW,
      THCudaTensor_data(state, output_n)
    );

    // Do Bias after:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m_ = nOutputPlane;
    long n_ = outputHeight * outputWidth;
    long k_ = 1;

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    THCudaBlas_gemm(
        state,
        't', 'n',
        n_, m_, k_,
        1,
        THCudaTensor_data(state, ones), k_,
        THCudaTensor_data(state, bias), k_,
        1,
        THCudaTensor_data(state, output_n), n_
    );

  }

  // Free
  THCudaTensor_free(state, input_n);
  THCudaTensor_free(state, output_n);

  // Resize output
  if (batch == 0) {
    THCudaTensor_resize3d(state, output, nOutputPlane, outputHeight, outputWidth);
    THCudaTensor_resize3d(state, input, nInputPlane, inputHeight, inputWidth);
  }

  // return output
  return 1;
}

static int cunn_SpatialDeconvolution_updateGradInput(lua_State *L) {
  THCState *state = getCutorchState(L);
  // Inputs
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

  // Params
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int nOutputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");
  int nInputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
  int padW = luaT_getfieldcheckint(L, 1, "padW");
  int padH = luaT_getfieldcheckint(L, 1, "padH");

  THCudaTensor *weight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "weight", "torch.CudaTensor");
  THCudaTensor *gradColumns = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "finput", "torch.CudaTensor");
  THCudaTensor *gradInput = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradInput", "torch.CudaTensor");

  THAssert(THCudaTensor_checkGPU(state, 5, input, gradOutput, weight,
                                 gradColumns, gradInput));
  luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch mode) tensor is expected");

  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCudaTensor_resize4d(state, input, 1, input->size[0], input->size[1], input->size[2]);
    THCudaTensor_resize4d(state, gradOutput, 1, gradOutput->size[0], gradOutput->size[1], gradOutput->size[2]);
  }

  long inputWidth   = input->size[3];
  long inputHeight  = input->size[2];
  long outputWidth  = (inputWidth - 1) * dW - 2*padW + kW;
  long outputHeight = (inputHeight - 1) * dH - 2*padH + kH;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Resize output
  THCudaTensor_resize4d(state, gradInput, batchSize, nInputPlane, inputHeight, inputWidth);

  // Resize temporary columns
  THCudaTensor_resize2d(state, gradColumns, nOutputPlane*kW*kH, inputHeight*inputWidth);

  // Helpers
  THCudaTensor *gradInput_n = THCudaTensor_new(state);
  THCudaTensor *gradOutput_n = THCudaTensor_new(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per sample:
    THCudaTensor_select(state, gradInput_n, gradInput, 0, elt);
    THCudaTensor_select(state, gradOutput_n, gradOutput, 0, elt);

    // Extract columns:
    im2col(
      THCState_getCurrentStream(state),
      THCudaTensor_data(state, gradOutput_n),
      nOutputPlane, outputHeight, outputWidth, kH, kW, padH, padW, dH, dW,
      THCudaTensor_data(state, gradColumns)
    );


    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = weight->size[0];
    long n = gradColumns->size[1];
    long k = weight->size[1];

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    THCudaBlas_gemm(
        state,
        'n', 'n',
        n, m, k,
	1,
        THCudaTensor_data(state, gradColumns), n,
        THCudaTensor_data(state, weight), k,
        0,
        THCudaTensor_data(state, gradInput_n), n
    );
  }


  // Free
  THCudaTensor_free(state, gradInput_n);
  THCudaTensor_free(state, gradOutput_n);

  // Resize output
  if (batch == 0) {
    THCudaTensor_resize3d(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCudaTensor_resize3d(state, input, nInputPlane, inputHeight, inputWidth);
    THCudaTensor_resize3d(state, gradInput, nInputPlane, inputHeight, inputWidth);
  }

  // Return gradInput
  return 1;
}


static int cunn_SpatialDeconvolution_accGradParameters(lua_State *L) {
  THCState *state = getCutorchState(L);
  // Inputs
  THCudaTensor *input = (THCudaTensor *)luaT_checkudata(L, 2, "torch.CudaTensor");
  THCudaTensor *gradOutput = (THCudaTensor *)luaT_checkudata(L, 3, "torch.CudaTensor");

  // Params
  int dW = luaT_getfieldcheckint(L, 1, "dW");
  int dH = luaT_getfieldcheckint(L, 1, "dH");
  int kW = luaT_getfieldcheckint(L, 1, "kW");
  int kH = luaT_getfieldcheckint(L, 1, "kH");
  int nOutputPlane = luaT_getfieldcheckint(L, 1, "nInputPlane");
  int nInputPlane = luaT_getfieldcheckint(L, 1, "nOutputPlane");
  int padW = luaT_getfieldcheckint(L, 1, "padW");
  int padH = luaT_getfieldcheckint(L, 1, "padH");
  float scale = luaL_optnumber(L, 4, 1);

  THCudaTensor *gradWeight = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradWeight", "torch.CudaTensor");
  THCudaTensor *gradBias = (THCudaTensor *)luaT_getfieldcheckudata(L, 1, "gradBias", "torch.CudaTensor");
  THCudaTensor *columns = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "finput", "torch.CudaTensor");
  THCudaTensor *ones = (THCudaTensor*)luaT_getfieldcheckudata(L, 1, "fgradInput", "torch.CudaTensor");

  THAssert(THCudaTensor_checkGPU(state, 6, input, gradOutput, gradWeight,
                                 gradBias, columns, ones));
  luaL_argcheck(L, input->nDimension == 3 || input->nDimension == 4, 2, "3D or 4D (batch mode) tensor is expected");

  int batch = 1;
  if (input->nDimension == 3) {
    // Force batch
    batch = 0;
    THCudaTensor_resize4d(state, input, 1, input->size[0], input->size[1], input->size[2]);
    THCudaTensor_resize4d(state, gradOutput, 1, gradOutput->size[0], gradOutput->size[1], gradOutput->size[2]);
  }

  long inputWidth   = input->size[3];
  long inputHeight  = input->size[2];
  long outputWidth  = (inputWidth - 1) * dW - 2*padW + kW;
  long outputHeight = (inputHeight - 1) * dH - 2*padH + kH;

  // Batch size + input planes
  long batchSize = input->size[0];

  // Define a buffer of ones, for bias accumulation
  if (ones->nDimension != 2 || ones->size[0]*ones->size[1] < outputHeight*outputWidth) {
    // Resize plane and fill with ones...
    THCudaTensor_resize2d(state, ones, outputHeight, outputWidth);
    THCudaTensor_fill(state, ones, 1);
  }

  // Resize temporary columns
  THCudaTensor_resize2d(state, columns, nOutputPlane*kW*kH, inputHeight*inputWidth);

  // Helpers
  THCudaTensor *input_n = THCudaTensor_new(state);
  THCudaTensor *gradOutput_n = THCudaTensor_new(state);

  // For each elt in batch, do:
  for (int elt = 0; elt < batchSize; elt ++) {
    // Matrix mulitply per output:
    THCudaTensor_select(state, input_n, input, 0, elt);
    THCudaTensor_select(state, gradOutput_n, gradOutput, 0, elt);

    // Extract columns:
    im2col(
      THCState_getCurrentStream(state),
      THCudaTensor_data(state, input_n),
      nOutputPlane, outputHeight, outputWidth, kH, kW, padH, padW, dH, dW,
      THCudaTensor_data(state, columns)
    );

    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m = gradWeight->size[0];
    long n = gradWeight->size[1];
    long k = columns->size[1];

    // Do GEMM (note: this is a bit confusing because gemm assumes column-major matrices)
    THCudaBlas_gemm(
        state,
        't', 'n',
        n, m, k,
        scale,
        THCudaTensor_data(state, columns), k,
        THCudaTensor_data(state, gradOutput_n), k,
        1,
        THCudaTensor_data(state, gradWeight), n
    );

    // Do Bias:
    // M,N,K are dims of matrix A and B
    // (see http://docs.nvidia.com/cuda/cublas/#cublas-lt-t-gt-gemm)
    long m_ = nOutputPlane;
    long k_ = outputHeight * outputWidth;

    // Do GEMV (note: this is a bit confusing because gemv assumes column-major matrices)
    THCudaBlas_gemv(
        state,
        't',
        k_, m_,
        scale,
        THCudaTensor_data(state, gradOutput_n), k_,
        THCudaTensor_data(state, ones), 1,
        1,
        THCudaTensor_data(state, gradBias), 1
    );
  }

  // Free
  THCudaTensor_free(state, input_n);
  THCudaTensor_free(state, gradOutput_n);

  // Resize
  if (batch == 0) {
    THCudaTensor_resize3d(state, gradOutput, nOutputPlane, outputHeight, outputWidth);
    THCudaTensor_resize3d(state, input, nInputPlane, inputHeight, inputWidth);
  }

  // Return nothing
  return 0;
}

const struct luaL_Reg cunn_SpatialDeconvolution__ [] = {
  {"SpatialDeconvolution_updateOutput", cunn_SpatialDeconvolution_updateOutput},
  {"SpatialDeconvolution_updateGradInput", cunn_SpatialDeconvolution_updateGradInput},
  {"SpatialDeconvolution_accGradParameters", cunn_SpatialDeconvolution_accGradParameters},
  {NULL, NULL}
};

void cunn_SpatialDeconvolution_init(lua_State *L)
{
  luaT_pushmetatable(L, "torch.CudaTensor");
  luaT_registeratname(L, cunn_SpatialDeconvolution__, "nn");
  lua_pop(L,1);
}

